#include "BHaH_defines.h"
#include "BHaH_gpu_defines.h"
#include "BHaH_gpu_function_prototypes.h"
__host__
void set_fd_constants() {
    const REAL h_FDPart1_Rational_1_2 = 1.0F / 2.0F;
    const REAL h_FDPart1_Rational_1_4 = 1.0F / 4.0F;
    const REAL h_FDPart1_Rational_1_12 = 1.0F / 12.0F;
    const REAL h_FDPart1_Rational_1_18 = 1.0F / 18.0F;
    const REAL h_FDPart1_Rational_1_144 = 1.0F / 144.0F;
    const REAL h_FDPart1_Rational_2_3 = 2.0F / 3.0F;    
    const REAL h_FDPart1_Rational_3_2 = 3.0F / 2.0F;
    const REAL h_FDPart1_Rational_4_3 = 4.0F / 3.0F;
    const REAL h_FDPart1_Rational_4_9 = 4.0F / 9.0F;
    const REAL h_FDPart1_Rational_5_2 = 5.0F / 2.0F;
    const REAL h_FDPart1_Rational_5_6 = 5.0F / 6.0F;

    hipMemcpyToSymbol(HIP_SYMBOL(FDPart1_Rational_1_2),&h_FDPart1_Rational_1_2, sizeof(REAL));
    hipMemcpyToSymbol(HIP_SYMBOL(FDPart1_Rational_1_4),&h_FDPart1_Rational_1_4, sizeof(REAL));
    hipMemcpyToSymbol(HIP_SYMBOL(FDPart1_Rational_1_12), &h_FDPart1_Rational_1_12, sizeof(REAL));
    hipMemcpyToSymbol(HIP_SYMBOL(FDPart1_Rational_1_18), &h_FDPart1_Rational_1_18, sizeof(REAL));
    hipMemcpyToSymbol(HIP_SYMBOL(FDPart1_Rational_1_144),&h_FDPart1_Rational_1_144, sizeof(REAL));
    hipMemcpyToSymbol(HIP_SYMBOL(FDPart1_Rational_2_3),  &h_FDPart1_Rational_2_3, sizeof(REAL));
    hipMemcpyToSymbol(HIP_SYMBOL(FDPart1_Rational_3_2),&h_FDPart1_Rational_3_2, sizeof(REAL));
    hipMemcpyToSymbol(HIP_SYMBOL(FDPart1_Rational_4_3),  &h_FDPart1_Rational_4_3, sizeof(REAL));
    hipMemcpyToSymbol(HIP_SYMBOL(FDPart1_Rational_4_9),  &h_FDPart1_Rational_4_9, sizeof(REAL));
    hipMemcpyToSymbol(HIP_SYMBOL(FDPart1_Rational_5_2),  &h_FDPart1_Rational_5_2, sizeof(REAL));
    hipMemcpyToSymbol(HIP_SYMBOL(FDPart1_Rational_5_6),&h_FDPart1_Rational_5_6, sizeof(REAL));
}

__host__
void set_param_constants(const params_struct *restrict params) {
    hipMemcpyToSymbol(HIP_SYMBOL(d_params), params, sizeof(params_struct));
}
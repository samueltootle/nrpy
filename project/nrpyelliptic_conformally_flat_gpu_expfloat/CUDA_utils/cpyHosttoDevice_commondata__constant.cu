#include "../BHaH_defines.h"
/*
 * Copy parameters to GPU __constant__.
 */
__host__ void cpyHosttoDevice_commondata__constant(const commondata_struct *restrict commondata) {
  hipMemcpyToSymbol(HIP_SYMBOL(d_commondata), commondata, sizeof(commondata_struct));
}

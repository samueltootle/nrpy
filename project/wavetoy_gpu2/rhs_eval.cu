#include "hip/hip_runtime.h"
#include "BHaH_defines.h"
#include "BHaH_gpu_defines.h"
#include "BHaH_gpu_function_prototypes.h"
#include <stdexcept>
/*
 * Set RHSs for wave equation.
 */
__global__
void rhs_eval_gpu(const commondata_struct *restrict commondata, 
              const params_struct *restrict params, 
              const REAL *restrict in_gfs,
              REAL *restrict rhs_gfs) {

// #include "set_CodeParameters.h"
  const REAL & invdxx0 = d_params.invdxx0;
  const REAL & invdxx1 = d_params.invdxx1;
  const REAL & invdxx2 = d_params.invdxx2;

  const int & Nxx0 = d_params.Nxx0;
  const int & Nxx1 = d_params.Nxx1;
  const int & Nxx2 = d_params.Nxx2;

  const int & Nxx_plus_2NGHOSTS0 = d_params.Nxx_plus_2NGHOSTS0;
  const int & Nxx_plus_2NGHOSTS1 = d_params.Nxx_plus_2NGHOSTS1;
  const int & Nxx_plus_2NGHOSTS2 = d_params.Nxx_plus_2NGHOSTS2;

  const int tid0  = blockIdx.x * blockDim.x + threadIdx.x;
  const int tid1  = blockIdx.y * blockDim.y + threadIdx.y;
  const int tid2  = blockIdx.z * blockDim.z + threadIdx.z;
  
  const int stride0 = blockDim.x * gridDim.x;
  const int stride1 = blockDim.y * gridDim.y;
  const int stride2 = blockDim.z * gridDim.z;
  // if(tid0 == 0 && tid1 == 0 && tid2 == 0) {
  //   printf("%f %f %f %u %u %u %u %u %u", 
  //     invdxx0,invdxx1,invdxx2,
  //     Nxx0,Nxx2,Nxx2,
  //     Nxx_plus_2NGHOSTS0,Nxx_plus_2NGHOSTS0,Nxx_plus_2NGHOSTS0);
  // }
  for (int i2 = tid2+NGHOSTS; i2 < NGHOSTS + Nxx2; i2+=stride2) {
    for (int i1 = tid1+NGHOSTS; i1 < NGHOSTS + Nxx1; i1+=stride1) {
      for (int i0 = tid0+NGHOSTS; i0 < NGHOSTS + Nxx0; i0+=stride0) {
        /*
         * NRPy+-Generated GF Access/FD Code, Step 1 of 2:
         * Read gridfunction(s) from main memory and compute FD stencils as needed.
         */
        const REAL uu_i2m2 = in_gfs[IDX4(UUGF, i0, i1, i2 - 2)];
        const REAL uu_i2m1 = in_gfs[IDX4(UUGF, i0, i1, i2 - 1)];
        const REAL uu_i1m2 = in_gfs[IDX4(UUGF, i0, i1 - 2, i2)];
        const REAL uu_i1m1 = in_gfs[IDX4(UUGF, i0, i1 - 1, i2)];
        const REAL uu_i0m2 = in_gfs[IDX4(UUGF, i0 - 2, i1, i2)];
        const REAL uu_i0m1 = in_gfs[IDX4(UUGF, i0 - 1, i1, i2)];
        const REAL uu = in_gfs[IDX4(UUGF, i0, i1, i2)];
        const REAL uu_i0p1 = in_gfs[IDX4(UUGF, i0 + 1, i1, i2)];
        const REAL uu_i0p2 = in_gfs[IDX4(UUGF, i0 + 2, i1, i2)];
        const REAL uu_i1p1 = in_gfs[IDX4(UUGF, i0, i1 + 1, i2)];
        const REAL uu_i1p2 = in_gfs[IDX4(UUGF, i0, i1 + 2, i2)];
        const REAL uu_i2p1 = in_gfs[IDX4(UUGF, i0, i1, i2 + 1)];
        const REAL uu_i2p2 = in_gfs[IDX4(UUGF, i0, i1, i2 + 2)];
        const REAL vv = in_gfs[IDX4(VVGF, i0, i1, i2)];
        
        // moved to __constant__ space
        // const REAL FDPart1_Rational_5_2 = 5.0 / 2.0;
        // const REAL FDPart1_Rational_1_12 = 1.0 / 12.0;
        // const REAL FDPart1_Rational_4_3 = 4.0 / 3.0;
        // printf("%f - %f - %f \n", FDPart1_Rational_5_2, FDPart1_Rational_1_12, FDPart1_Rational_4_3); 
        // printf("%f \n", wavespeed);
        
        const REAL FDPart1tmp0 = -FDPart1_Rational_5_2 * uu;
        const REAL uu_dDD00 =
            ((invdxx0) * (invdxx0)) * (FDPart1_Rational_1_12 * (-uu_i0m2 - uu_i0p2) + FDPart1_Rational_4_3 * (uu_i0m1 + uu_i0p1) + FDPart1tmp0);
        const REAL uu_dDD11 =
            ((invdxx1) * (invdxx1)) * (FDPart1_Rational_1_12 * (-uu_i1m2 - uu_i1p2) + FDPart1_Rational_4_3 * (uu_i1m1 + uu_i1p1) + FDPart1tmp0);
        const REAL uu_dDD22 =
            ((invdxx2) * (invdxx2)) * (FDPart1_Rational_1_12 * (-uu_i2m2 - uu_i2p2) + FDPart1_Rational_4_3 * (uu_i2m1 + uu_i2p1) + FDPart1tmp0);

        /*
         * NRPy+-Generated GF Access/FD Code, Step 2 of 2:
         * Evaluate SymPy expressions and write to main memory.
         */
        const REAL FDPart3tmp0 = ((wavespeed) * (wavespeed));
        rhs_gfs[IDX4(UUGF, i0, i1, i2)] = vv;
        rhs_gfs[IDX4(VVGF, i0, i1, i2)] = FDPart3tmp0 * uu_dDD00 + FDPart3tmp0 * uu_dDD11 + FDPart3tmp0 * uu_dDD22;

      } // END LOOP: for (int i0 = NGHOSTS; i0 < NGHOSTS+Nxx0; i0++)
    }   // END LOOP: for (int i1 = NGHOSTS; i1 < NGHOSTS+Nxx1; i1++)
  }     // END LOOP: for (int i2 = NGHOSTS; i2 < NGHOSTS+Nxx2; i2++)
}

__global__ void compute_uu_dDDxx_gpu(const params_struct *restrict params, 
                                 const REAL *restrict in_gfs,
                                 REAL *restrict aux_gfs)
{ 

  extern __shared__ float s_f[]; // 2-wide halo for 4th order FD

  const REAL & invdxx0 = d_params.invdxx0;

  const int & Nxx0 = d_params.Nxx0;

  const int & Nxx_plus_2NGHOSTS0 = d_params.Nxx_plus_2NGHOSTS0;
  const int & Nxx_plus_2NGHOSTS1 = d_params.Nxx_plus_2NGHOSTS1;
  const int & Nxx_plus_2NGHOSTS2 = d_params.Nxx_plus_2NGHOSTS2;

  // Local tile indices - not global thread indicies
  int tid0  = threadIdx.x;
  int tid1  = blockIdx.x*blockDim.y + threadIdx.y;
  int tid2  = blockIdx.y;
  int si = tid0 + NGHOSTS; // local i for shared memory access + halo offset
  int sj = threadIdx.y; // local j for shared memory access
  
  // s_f stores pencils in linear memory so we need a
  // shared memory index
  int sm_idx = sj * Nxx0 + si;

  // Global memory index - need to shift by ghost zones
  int i = tid0 + NGHOSTS;
  int j = tid1 + NGHOSTS;
  int k = tid2 + NGHOSTS;
  int globalIdx = IDX4(UUGF, i, j, k);

  s_f[sm_idx] = in_gfs[globalIdx];

  __syncthreads();

  // fill in SM ghost zones
  if (tid0 < NGHOSTS) {
    int temp_idx = IDX4(UUGF, i-NGHOSTS, j, k);
    s_f[sm_idx - NGHOSTS] = in_gfs[temp_idx];
    
    temp_idx = IDX4(UUGF, i + Nxx0 + NGHOSTS, j, k);
    s_f[sm_idx+Nxx0] = in_gfs[temp_idx];
  }

  __syncthreads();
  const REAL uu_i0m2 = s_f[sm_idx - 2];
  const REAL uu_i0m1 = s_f[sm_idx - 1];
  const REAL uu      = s_f[sm_idx    ];
  const REAL uu_i0p1 = s_f[sm_idx + 1];
  const REAL uu_i0p2 = s_f[sm_idx + 2];

  const REAL FDPart1tmp0 = -FDPart1_Rational_5_2 * uu;

  int globalIdx_out = IDX4(UD00, i, j, k);
  aux_gfs[globalIdx_out] = ((invdxx0) * (invdxx0)) * (
      FDPart1_Rational_1_12 * (-uu_i0m2 - uu_i0p2) 
    + FDPart1_Rational_4_3 * (uu_i0m1 + uu_i0p1) 
    + FDPart1tmp0
  );
}

__global__ void compute_uu_dDDyy_gpu(const params_struct *restrict params, 
                                 const REAL *restrict in_gfs,
                                 REAL *restrict aux_gfs)
{ 

  extern __shared__ float s_f[]; // 2-wide halo for 4th order FD

  const REAL & invdxx1 = d_params.invdxx1;

  // const int & Nxx0 = d_params.Nxx0;
  const int & Nxx1 = d_params.Nxx1;

  const int & Nxx_plus_2NGHOSTS0 = d_params.Nxx_plus_2NGHOSTS0;
  const int & Nxx_plus_2NGHOSTS1 = d_params.Nxx_plus_2NGHOSTS1;
  const int & Nxx_plus_2NGHOSTS2 = d_params.Nxx_plus_2NGHOSTS2;

  // Local tile indices - not global thread indicies
  int tid0  = blockIdx.x*blockDim.x + threadIdx.x;
  int tid1  = threadIdx.y;
  int tid2  = blockIdx.y;
  int si = threadIdx.x; // local i for shared memory access

  // Global array indicies
  int i = tid0 + NGHOSTS;
  int k = tid2 + NGHOSTS;  

  for (int j = tid1 + NGHOSTS; j < Nxx1 + NGHOSTS; j += blockDim.y) {
    int sj = j;
    
    // s_f stores pencils in linear memory so we need a
    // shared memory index such that the contiguous elements
    // are now the "y" data.
    int sm_idx = si * Nxx1 + sj;

    int globalIdx = IDX4(UUGF, i, j, k);

    s_f[sm_idx] = aux_gfs[globalIdx];
  }

  int sj = tid1 + NGHOSTS;
  int sm_idx = si * Nxx1 + sj;
  int j = sj;
  __syncthreads();
  
  // fill in SM ghost zones
  if (tid1 < NGHOSTS) {
    uint temp_idx1 = IDX4(UUGF, i, j - NGHOSTS, k);
    s_f[sm_idx-NGHOSTS]  = in_gfs[temp_idx1];
    uint temp_idx2 = IDX4(UUGF, i, j + Nxx1 + NGHOSTS, k);
    // printf("%d - %d : %d - %d \n", temp_idx1, sm_idx-4, temp_idx2, sm_idx + Nxx1 + NGHOSTS);
    s_f[sm_idx+Nxx1+NGHOSTS] = in_gfs[temp_idx2];
  }

  __syncthreads();

  for (int j = tid1 + NGHOSTS; j < Nxx1 + NGHOSTS; j += blockDim.y) {
    int sj = j;
    int sm_idx = si * Nxx1 + sj;

    const REAL uu_j0m2 = s_f[sm_idx - 2];
    const REAL uu_j0m1 = s_f[sm_idx - 1];
    const REAL uu      = s_f[sm_idx    ];
    const REAL uu_j0p1 = s_f[sm_idx + 1];
    const REAL uu_j0p2 = s_f[sm_idx + 2];

    const REAL FDPart1tmp0 = -FDPart1_Rational_5_2 * uu;

    int globalIdx_out = IDX4(UD11, i, j, k);
    aux_gfs[globalIdx_out] = ((invdxx1) * (invdxx1)) * (
        FDPart1_Rational_1_12 * (-uu_j0m2 - uu_j0p2) 
      + FDPart1_Rational_4_3 * (uu_j0m1 + uu_j0p1) 
      + FDPart1tmp0
    );
  }
}

__global__ void compute_uu_dDDzz_gpu(const params_struct *restrict params, 
                                 const REAL *restrict in_gfs,
                                 REAL *restrict aux_gfs)
{ 

  extern __shared__ float s_f[]; // 2-wide halo for 4th order FD

  const REAL & invdxx2 = d_params.invdxx2;

  // const int & Nxx0 = d_params.Nxx0;
  const int & Nxx2 = d_params.Nxx2;

  const int & Nxx_plus_2NGHOSTS0 = d_params.Nxx_plus_2NGHOSTS0;
  const int & Nxx_plus_2NGHOSTS1 = d_params.Nxx_plus_2NGHOSTS1;
  const int & Nxx_plus_2NGHOSTS2 = d_params.Nxx_plus_2NGHOSTS2;

  // Local tile indices - not global thread indicies
  int tid0  = blockIdx.x*blockDim.x + threadIdx.x;
  int tid1  = threadIdx.y;
  int tid2  = blockIdx.y;
  int si = threadIdx.x; // local i for shared memory access

  // Global array indicies
  int i = tid0 + NGHOSTS;
  int j = tid2 + NGHOSTS;  

  for (int k = tid1 + NGHOSTS; k < Nxx2 + NGHOSTS; k += blockDim.y) {
    int sk = k;
    
    // s_f stores pencils in linear memory so we need a
    // shared memory index such that the contiguous elements
    // are now the "y" data.
    int sm_idx = si * Nxx2 + sk;

    int globalIdx = IDX4(UUGF, i, j, k);

    s_f[sm_idx] = aux_gfs[globalIdx];
  }
  __syncthreads();
  
  // fill in SM ghost zones
  if (tid1 < NGHOSTS) {
    int sk = tid1 + NGHOSTS;
    int sm_idx = si * Nxx2 + sk;
    int k = sk;
    uint temp_idx1 = IDX4(UUGF, i, j, k - NGHOSTS);
    s_f[sm_idx-NGHOSTS]  = in_gfs[temp_idx1];
    uint temp_idx2 = IDX4(UUGF, i, j, k + Nxx2 + NGHOSTS);
    // printf("%d - %d : %d - %d \n", temp_idx1, sm_idx-4, temp_idx2, sm_idx + Nxx1 + NGHOSTS);
    s_f[sm_idx + Nxx2 + NGHOSTS] = in_gfs[temp_idx2];
  }

  __syncthreads();

  for (int k = tid1 + NGHOSTS; k < Nxx2 + NGHOSTS; k += blockDim.y) {
    int sk = k;
    int sm_idx = si * Nxx2 + sk;

    const REAL uu_j0m2 = s_f[sm_idx - 2];
    const REAL uu_j0m1 = s_f[sm_idx - 1];
    const REAL uu      = s_f[sm_idx    ];
    const REAL uu_j0p1 = s_f[sm_idx + 1];
    const REAL uu_j0p2 = s_f[sm_idx + 2];

    const REAL FDPart1tmp0 = -FDPart1_Rational_5_2 * uu;

    int globalIdx_out = IDX4(UD22, i, j, k);
    aux_gfs[globalIdx_out] = ((invdxx2) * (invdxx2)) * (
        FDPart1_Rational_1_12 * (-uu_j0m2 - uu_j0p2) 
      + FDPart1_Rational_4_3 * (uu_j0m1 + uu_j0p1) 
      + FDPart1tmp0
    );
  }
}

__host__ 
void compute_uu_dDDxx(const params_struct *restrict params, 
                          const REAL *restrict in_gfs,
                          REAL *restrict aux_gfs,
                          const int Nxx0,
                          const int Nxx1,
                          const int Nxx2,
                          const int Nxx_plus_2NGHOSTS0) {
  // for dx, we allocated pencils of threads that are 
  // Nxx0 x PENCIL_SIZEY dimensions per block
  dim3 block_threads(Nxx0, PENCIL_SIZEY, 1);
  
  // We have Nxx2 x (Nxx1 / PENCIL_SIZEY) blocks
  dim3 grid_blocks(Nxx1 / PENCIL_SIZEY, Nxx2, 1);
  
  // hipMemset(aux_gfs, 0, NUM_AUXEVOL_GFS * sizeof(REAL));
  // cudaCheckErrors(hipMemset, "memory failed")

  // Determine dynamic shared memory size in bytes
  const size_t SM_size = PENCIL_SIZEY * Nxx_plus_2NGHOSTS0 * sizeof(REAL);
  
  // Fetch maximum shared memory size per block
  const int device = 0; //assumes single GPU
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, device);
  const size_t maxMemPerBlock = deviceProp.sharedMemPerBlock;

  // printf("SM_size : %lu , max: %lu\n\n\n", SM_size, maxMemPerBlock);
  
  if(SM_size > maxMemPerBlock) {
    throw std::runtime_error("Grid is too large for current GPU shared memory restrictions");
  }

  compute_uu_dDDxx_gpu<<<grid_blocks, block_threads, SM_size>>>(params, in_gfs, aux_gfs);
  cudaCheckErrors(compute_uu_dDDxx_gpu, "kernel failed")
}

__host__ 
void compute_uu_dDDyy(const params_struct *restrict params, 
                          const REAL *restrict in_gfs,
                          REAL *restrict aux_gfs,
                          const int Nxx0,
                          const int Nxx1,
                          const int Nxx2,
                          const int Nxx_plus_2NGHOSTS1) {
  // To ensure coalescence, we want retain reads in the x-direction
  // i.e. the contiguous memory space, based on the standard
  // instruction limits.  Each warp will attempt memory reads up
  // to 128 bytes in a single instruction, in powers of 2,
  // (e.g. 8,16,32,64,128), but this is dependent on the 
  // compute capability of the GPU.  Here we dedicate
  // one thread per data element we read in the x-direction.
  size_t threads_in_x_dir = 128 / sizeof(REAL);

  // Max threads in the y-direction.  Even if we can read
  // the entire tile into shared memory, that doesn't mean
  // we have enough threads per SM to process the entire tile.
  // Therefore we can only have a maximum number of threads in the
  // y direction and each thread will have to compute multiple points.
  size_t threads_in_y_dir = 1024 / threads_in_x_dir;

  // The tile size should attempt to avoid halo data,
  // i.e. zones of data that are read by two or more blocks
  // into shared memory.
  // For the simple case of cartesian coordinates, the
  // derivatives are 1D, so this shouldn't be a problem
  // so long as the 1D Grid size * threads_in_x_dir
  // will fit into shared memory.
  // Determine dynamic shared memory size in bytes:
  const size_t SM_size = threads_in_x_dir * Nxx_plus_2NGHOSTS1 * sizeof(REAL);
  
  // For now we just throw an exception based on maximum shared memory size per block
  // but it should be possible to decide a better course of action - e.g. using halos
  // instead
  const int device = 0; //assumes single GPU
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, device);
  const size_t maxMemPerBlock = deviceProp.sharedMemPerBlock;
  if(SM_size > maxMemPerBlock) {
    throw std::runtime_error("Grid is too large for current GPU shared memory restrictions");
  }

  // Setup our thread layout
  dim3 block_threads(threads_in_x_dir, threads_in_y_dir, 1);
  
  // Setup our grid layout such that our tiles will iterate through the entire
  // numerical space
  dim3 grid_blocks(Nxx0 / threads_in_x_dir, Nxx2, 1);

  // printf("SM_size : %lu , max: %lu\n\n\n", SM_size, maxMemPerBlock);
  
  compute_uu_dDDyy_gpu<<<grid_blocks, block_threads, SM_size>>>(params, in_gfs, aux_gfs);
  cudaCheckErrors(compute_uu_dDDyy_gpu, "kernel failed")
  // hipDeviceSynchronize();
  // printf("SM lengh: %u\n",threads_in_x_dir * Nxx_plus_2NGHOSTS1);
}

__host__ 
void compute_uu_dDDzz(const params_struct *restrict params, 
                          const REAL *restrict in_gfs,
                          REAL *restrict aux_gfs,
                          const int Nxx0,
                          const int Nxx1,
                          const int Nxx2,
                          const int Nxx_plus_2NGHOSTS2) {
  // To ensure coalescence, we want retain reads in the x-direction
  // i.e. the contiguous memory space, based on the standard
  // instruction limits.  Each warp will attempt memory reads up
  // to 128 bytes in a single instruction, in powers of 2,
  // (e.g. 8,16,32,64,128), but this is dependent on the 
  // compute capability of the GPU.  Here we dedicate
  // one thread per data element we read in the x-direction.
  size_t threads_in_x_dir = 128 / sizeof(REAL);

  // Max threads in the y-direction.  Even if we can read
  // the entire tile into shared memory, that doesn't mean
  // we have enough threads per SM to process the entire tile.
  // Therefore we can only have a maximum number of threads in the
  // y direction and each thread will have to compute multiple points.
  size_t threads_in_z_dir = 1024 / threads_in_x_dir;

  // The tile size should attempt to avoid halo data,
  // i.e. zones of data that are read by two or more blocks
  // into shared memory.
  // For the simple case of cartesian coordinates, the
  // derivatives are 1D, so this shouldn't be a problem
  // so long as the 1D Grid size * threads_in_x_dir
  // will fit into shared memory.
  // Determine dynamic shared memory size in bytes:
  const size_t SM_size = threads_in_x_dir * Nxx_plus_2NGHOSTS2 * sizeof(REAL);
  
  // For now we just throw an exception based on maximum shared memory size per block
  // but it should be possible to decide a better course of action - e.g. using halos
  // instead
  const int device = 0; //assumes single GPU
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, device);
  const size_t maxMemPerBlock = deviceProp.sharedMemPerBlock;
  if(SM_size > maxMemPerBlock) {
    throw std::runtime_error("Grid is too large for current GPU shared memory restrictions");
  }

  // Setup our thread layout
  dim3 block_threads(threads_in_x_dir, threads_in_z_dir, 1);
  
  // Setup our grid layout such that our tiles will iterate through the entire
  // numerical space
  dim3 grid_blocks(Nxx0 / threads_in_x_dir, Nxx1, 1);

  // printf("SM_size : %lu , max: %lu\n\n\n", SM_size, maxMemPerBlock);
  
  compute_uu_dDDzz_gpu<<<grid_blocks, block_threads, SM_size>>>(params, in_gfs, aux_gfs);
  cudaCheckErrors(compute_uu_dDDzz_gpu, "kernel failed")
}

__host__
void rhs_eval(const commondata_struct *restrict commondata, 
              const params_struct *restrict params, 
              const REAL *restrict in_gfs,
              REAL *restrict rhs_gfs,
              REAL *restrict aux_gfs) {
#ifdef ORIG_RHS
  int Nxx_plus_2NGHOSTS0, Nxx_plus_2NGHOSTS1, Nxx_plus_2NGHOSTS2;
  hipMemcpy(&Nxx_plus_2NGHOSTS0, &params->Nxx_plus_2NGHOSTS0, sizeof(int), hipMemcpyDeviceToHost);
  cudaCheckErrors(hipMemcpy, "memory failed")
  hipMemcpy(&Nxx_plus_2NGHOSTS1, &params->Nxx_plus_2NGHOSTS1, sizeof(int), hipMemcpyDeviceToHost);
  cudaCheckErrors(hipMemcpy, "memory failed")
  hipMemcpy(&Nxx_plus_2NGHOSTS2, &params->Nxx_plus_2NGHOSTS2, sizeof(int), hipMemcpyDeviceToHost);
  cudaCheckErrors(hipMemcpy, "memory failed")
  dim3 block(GPU_NBLOCK0,GPU_NBLOCK1,GPU_NBLOCK2);
  dim3 grid(
    (Nxx_plus_2NGHOSTS0 + GPU_NBLOCK0 - 1) / GPU_NBLOCK0,
    (Nxx_plus_2NGHOSTS1 + GPU_NBLOCK1 - 1) / GPU_NBLOCK1,
    (Nxx_plus_2NGHOSTS2 + GPU_NBLOCK2 - 1) / GPU_NBLOCK2
  );
  rhs_eval_gpu<<<grid,block>>>(commondata, params, in_gfs, rhs_gfs);
  cudaCheckErrors(rhs_eval_gpu, "kernel failed")
  testcpy(in_gfs);
#endif
  // Nxx per coordinate direction
  int Nxx0, Nxx1, Nxx2;
  hipMemcpy(&Nxx0, &params->Nxx0, sizeof(int), hipMemcpyDeviceToHost);
  cudaCheckErrors(hipMemcpy, "memory failed")
  hipMemcpy(&Nxx1, &params->Nxx1, sizeof(int), hipMemcpyDeviceToHost);
  cudaCheckErrors(hipMemcpy, "memory failed")
  hipMemcpy(&Nxx2, &params->Nxx2, sizeof(int), hipMemcpyDeviceToHost);
  cudaCheckErrors(hipMemcpy, "memory failed")
  
  // Nxx + ghost zones per coordinate direction
  int Nxx_plus_2NGHOSTS0, Nxx_plus_2NGHOSTS1, Nxx_plus_2NGHOSTS2;
  hipMemcpy(&Nxx_plus_2NGHOSTS0, &params->Nxx_plus_2NGHOSTS0, sizeof(int), hipMemcpyDeviceToHost);
  cudaCheckErrors(hipMemcpy, "memory failed")
  hipMemcpy(&Nxx_plus_2NGHOSTS1, &params->Nxx_plus_2NGHOSTS1, sizeof(int), hipMemcpyDeviceToHost);
  cudaCheckErrors(hipMemcpy, "memory failed")
  hipMemcpy(&Nxx_plus_2NGHOSTS2, &params->Nxx_plus_2NGHOSTS2, sizeof(int), hipMemcpyDeviceToHost);
  cudaCheckErrors(hipMemcpy, "memory failed")
  
  compute_uu_dDDxx(params, in_gfs, aux_gfs, Nxx0, Nxx1, Nxx2,Nxx_plus_2NGHOSTS0);
  compute_uu_dDDyy(params, in_gfs, aux_gfs, Nxx0, Nxx1, Nxx2,Nxx_plus_2NGHOSTS1);
  compute_uu_dDDzz(params, in_gfs, aux_gfs, Nxx0, Nxx1, Nxx2,Nxx_plus_2NGHOSTS2);

  // compute_rhs(params, in_gfs, aux_gfs, Nxx0, Nxx1, Nxx2,Nxx_plus_2NGHOSTS2);

  // dim3 block(GPU_NBLOCK0,GPU_NBLOCK1,GPU_NBLOCK2);
  // dim3 grid(
  //   (Nxx_plus_2NGHOSTS0 + GPU_NBLOCK0 - 1) / GPU_NBLOCK0,
  //   (Nxx_plus_2NGHOSTS1 + GPU_NBLOCK1 - 1) / GPU_NBLOCK1,
  //   (Nxx_plus_2NGHOSTS2 + GPU_NBLOCK2 - 1) / GPU_NBLOCK2
  // );
  // rhs_eval_gpu<<<grid,block>>>(commondata, params, in_gfs, rhs_gfs);
  // cudaCheckErrors(rhs_eval_gpu, "kernel failed")
}

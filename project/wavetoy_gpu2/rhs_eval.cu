#include "hip/hip_runtime.h"
#include "BHaH_defines.h"
#include "BHaH_gpu_defines.h"
#include "BHaH_gpu_function_prototypes.h"
#include <stdexcept>
#define DEBUG_INDEX 35114
/*
 * Set RHSs for wave equation.
 */
__global__
void rhs_eval_gpu(const commondata_struct *restrict commondata, 
              const params_struct *restrict params, 
              const REAL *restrict in_gfs,
              REAL *restrict rhs_gfs) {

// #include "set_CodeParameters.h"
  const REAL & invdxx0 = d_params.invdxx0;
  const REAL & invdxx1 = d_params.invdxx1;
  const REAL & invdxx2 = d_params.invdxx2;

  const int & Nxx0 = d_params.Nxx0;
  const int & Nxx1 = d_params.Nxx1;
  const int & Nxx2 = d_params.Nxx2;

  const int & Nxx_plus_2NGHOSTS0 = d_params.Nxx_plus_2NGHOSTS0;
  const int & Nxx_plus_2NGHOSTS1 = d_params.Nxx_plus_2NGHOSTS1;
  const int & Nxx_plus_2NGHOSTS2 = d_params.Nxx_plus_2NGHOSTS2;

  const int tid0  = blockIdx.x * blockDim.x + threadIdx.x;
  const int tid1  = blockIdx.y * blockDim.y + threadIdx.y;
  const int tid2  = blockIdx.z * blockDim.z + threadIdx.z;
  
  const int stride0 = blockDim.x * gridDim.x;
  const int stride1 = blockDim.y * gridDim.y;
  const int stride2 = blockDim.z * gridDim.z;
  // if(tid0 == 0 && tid1 == 0 && tid2 == 0) {
  //   printf("%f %f %f %u %u %u %u %u %u", 
  //     invdxx0,invdxx1,invdxx2,
  //     Nxx0,Nxx2,Nxx2,
  //     Nxx_plus_2NGHOSTS0,Nxx_plus_2NGHOSTS0,Nxx_plus_2NGHOSTS0);
  // }
  for (int i2 = tid2+NGHOSTS; i2 < NGHOSTS + Nxx2; i2+=stride2) {
    for (int i1 = tid1+NGHOSTS; i1 < NGHOSTS + Nxx1; i1+=stride1) {
      for (int i0 = tid0+NGHOSTS; i0 < NGHOSTS + Nxx0; i0+=stride0) {
        /*
         * NRPy+-Generated GF Access/FD Code, Step 1 of 2:
         * Read gridfunction(s) from main memory and compute FD stencils as needed.
         */
        const REAL uu_i2m2 = in_gfs[IDX4(UUGF, i0, i1, i2 - 2)];
        const REAL uu_i2m1 = in_gfs[IDX4(UUGF, i0, i1, i2 - 1)];
        const REAL uu_i1m2 = in_gfs[IDX4(UUGF, i0, i1 - 2, i2)];
        const REAL uu_i1m1 = in_gfs[IDX4(UUGF, i0, i1 - 1, i2)];
        const REAL uu_i0m2 = in_gfs[IDX4(UUGF, i0 - 2, i1, i2)];
        const REAL uu_i0m1 = in_gfs[IDX4(UUGF, i0 - 1, i1, i2)];
        const REAL uu = in_gfs[IDX4(UUGF, i0, i1, i2)];
        const REAL uu_i0p1 = in_gfs[IDX4(UUGF, i0 + 1, i1, i2)];
        const REAL uu_i0p2 = in_gfs[IDX4(UUGF, i0 + 2, i1, i2)];
        const REAL uu_i1p1 = in_gfs[IDX4(UUGF, i0, i1 + 1, i2)];
        const REAL uu_i1p2 = in_gfs[IDX4(UUGF, i0, i1 + 2, i2)];
        const REAL uu_i2p1 = in_gfs[IDX4(UUGF, i0, i1, i2 + 1)];
        const REAL uu_i2p2 = in_gfs[IDX4(UUGF, i0, i1, i2 + 2)];
        const REAL vv = in_gfs[IDX4(VVGF, i0, i1, i2)];
        
        // moved to __constant__ space
        // const REAL FDPart1_Rational_5_2 = 5.0 / 2.0;
        // const REAL FDPart1_Rational_1_12 = 1.0 / 12.0;
        // const REAL FDPart1_Rational_4_3 = 4.0 / 3.0;
        // printf("%f - %f - %f \n", FDPart1_Rational_5_2, FDPart1_Rational_1_12, FDPart1_Rational_4_3); 
        // printf("%f \n", wavespeed);
        
        const REAL FDPart1tmp0 = -FDPart1_Rational_5_2 * uu;
        const REAL uu_dDD00 =
            ((invdxx0) * (invdxx0)) * (FDPart1_Rational_1_12 * (-uu_i0m2 - uu_i0p2) + FDPart1_Rational_4_3 * (uu_i0m1 + uu_i0p1) + FDPart1tmp0);
        const REAL uu_dDD11 =
            ((invdxx1) * (invdxx1)) * (FDPart1_Rational_1_12 * (-uu_i1m2 - uu_i1p2) + FDPart1_Rational_4_3 * (uu_i1m1 + uu_i1p1) + FDPart1tmp0);
        const REAL uu_dDD22 =
            ((invdxx2) * (invdxx2)) * (FDPart1_Rational_1_12 * (-uu_i2m2 - uu_i2p2) + FDPart1_Rational_4_3 * (uu_i2m1 + uu_i2p1) + FDPart1tmp0);

        /*
         * NRPy+-Generated GF Access/FD Code, Step 2 of 2:
         * Evaluate SymPy expressions and write to main memory.
         */
        const REAL FDPart3tmp0 = ((wavespeed) * (wavespeed));
        rhs_gfs[IDX4(UUGF, i0, i1, i2)] = vv;
        rhs_gfs[IDX4(VVGF, i0, i1, i2)] = FDPart3tmp0 * uu_dDD00 + FDPart3tmp0 * uu_dDD11 + FDPart3tmp0 * uu_dDD22;
        #ifdef DEBUG_RHS
        if(IDX4(UUGF, i0, i1, i2) == DEBUG_INDEX) {
          printf("uD00: %1.15f - %1.15f - %1.15f - %1.15f - %1.15f - %1.15f\n"
                 "uD11: %1.15f - %1.15f - %1.15f - %1.15f - %1.15f - %1.15f\n"
                 "uD22: %1.15f - %1.15f - %1.15f - %1.15f - %1.15f - %1.15f\n"
                 "rhs:  %1.15f - %1.15f\n\n",
            uu_dDD00, uu_i0m2, uu_i0m1, uu, uu_i0p1, uu_i0p2, 
            uu_dDD11, uu_i1m2, uu_i1m1, uu, uu_i1p1, uu_i1p2,
            uu_dDD22, uu_i2m2, uu_i2m1, uu, uu_i2p1, uu_i2p2,
            rhs_gfs[IDX4(UUGF, i0, i1, i2)], rhs_gfs[IDX4(VVGF, i0, i1, i2)]);
          // printf("%u - %f - %f: %f - %f - %f\n", IDX4(UUGF, i0, i1, i2), rhs_gfs[IDX4(UUGF, i0, i1, i2)], uu, uu_dDD00, uu_dDD11, uu_dDD22);
        }
        #endif

      } // END LOOP: for (int i0 = NGHOSTS; i0 < NGHOSTS+Nxx0; i0++)
    }   // END LOOP: for (int i1 = NGHOSTS; i1 < NGHOSTS+Nxx1; i1++)
  }     // END LOOP: for (int i2 = NGHOSTS; i2 < NGHOSTS+Nxx2; i2++)
}

__global__ void compute_uu_dDDxx_gpu(const params_struct *restrict params, 
                                 const REAL *restrict in_gfs,
                                 REAL *restrict aux_gfs)
{ 

  extern __shared__ REAL s_f[]; // 2-wide halo for 4th order FD

  const REAL & invdxx0 = d_params.invdxx0;

  const int & Nxx0 = d_params.Nxx0;

  const int & Nxx_plus_2NGHOSTS0 = d_params.Nxx_plus_2NGHOSTS0;
  const int & Nxx_plus_2NGHOSTS1 = d_params.Nxx_plus_2NGHOSTS1;
  const int & Nxx_plus_2NGHOSTS2 = d_params.Nxx_plus_2NGHOSTS2;

  // Local tile indices - not global thread indicies
  int tid0  = threadIdx.x;
  int tid1  = blockIdx.x * blockDim.y + threadIdx.y;
  int tid2  = blockIdx.y;
  int si = tid0 + NGHOSTS; // local i for shared memory access + halo offset
  int sj = threadIdx.y; // local j for shared memory access
  
  // s_f stores pencils in linear memory so we need a
  // shared memory index
  int sm_idx = sj * Nxx_plus_2NGHOSTS0 + si;

  // Global memory index - need to shift by ghost zones
  int i = tid0 + NGHOSTS;
  int j = tid1 + NGHOSTS;
  int k = tid2 + NGHOSTS;
  int globalIdx = IDX4(UUGF, i, j, k);

  s_f[sm_idx] = in_gfs[globalIdx];

  __syncthreads();

  // fill in SM ghost zones
  if (tid0 < NGHOSTS) {
    int temp_idx = IDX4(UUGF, i - NGHOSTS, j, k);
    s_f[sm_idx - NGHOSTS] = in_gfs[temp_idx];
    
    temp_idx = IDX4(UUGF, i + Nxx0, j, k);
    s_f[sm_idx+Nxx0] = in_gfs[temp_idx];
  }

  __syncthreads();
  const REAL uu_i0m2 = s_f[sm_idx - 2];
  const REAL uu_i0m1 = s_f[sm_idx - 1];
  const REAL uu      = s_f[sm_idx    ];
  const REAL uu_i0p1 = s_f[sm_idx + 1];
  const REAL uu_i0p2 = s_f[sm_idx + 2];

  const REAL FDPart1tmp0 = -FDPart1_Rational_5_2 * uu;

  int globalIdx_out = IDX4(UD00, i, j, k);
  aux_gfs[globalIdx_out] = ((invdxx0) * (invdxx0)) * (
      FDPart1_Rational_1_12 * (-uu_i0m2 - uu_i0p2) 
    + FDPart1_Rational_4_3 * (uu_i0m1 + uu_i0p1) 
    + FDPart1tmp0
  );
  #ifdef DEBUG_RHS
  if(globalIdx == DEBUG_INDEX) {
    printf("uD00: %1.15f - %1.15f - %1.15f - %1.15f - %1.15f - %1.15f\n", 
      aux_gfs[globalIdx_out], uu_i0m2, uu_i0m1, uu, uu_i0p1, uu_i0p2);
  }
  #endif
}

__global__ void compute_uu_dDDyy_gpu(const params_struct *restrict params, 
                                 const REAL *restrict in_gfs,
                                 REAL *restrict aux_gfs)
{ 

  extern __shared__ REAL s_f[];

  const REAL & invdxx1 = d_params.invdxx1;

  // const int & Nxx0 = d_params.Nxx0;
  const int & Nxx1 = d_params.Nxx1;

  const int & Nxx_plus_2NGHOSTS0 = d_params.Nxx_plus_2NGHOSTS0;
  const int & Nxx_plus_2NGHOSTS1 = d_params.Nxx_plus_2NGHOSTS1;
  const int & Nxx_plus_2NGHOSTS2 = d_params.Nxx_plus_2NGHOSTS2;

  // Local tile indices - not global thread indicies
  int tid0  = blockIdx.x*blockDim.x + threadIdx.x;
  int tid1  = threadIdx.y;
  int tid2  = blockIdx.y;
  int si = threadIdx.x; // local i for shared memory access

  // Global array indicies
  int i = tid0 + NGHOSTS;
  int k = tid2 + NGHOSTS;

  for (int j = tid1 + NGHOSTS; j < Nxx1 + NGHOSTS; j += blockDim.y) {
    int sj = j;
    
    // s_f stores pencils in linear memory so we need a
    // shared memory index such that the contiguous elements
    // are now the "y" data.
    int sm_idx = si * Nxx_plus_2NGHOSTS1 + sj;

    int globalIdx = IDX4(UUGF, i, j, k);

    s_f[sm_idx] = in_gfs[globalIdx];
    // if(sm_idx == 2 && blockIdx.x == 0 && blockIdx.y == 0) {
    //   printf("sf: (%u, %u) %u, %1.15f - gf: %1.15f\n\n", 
    //     blockIdx.x, blockIdx.y, sm_idx, s_f[sm_idx], in_gfs[globalIdx]);
    // }
  }

  int sj = tid1 + NGHOSTS;
  int sm_idx = si * Nxx_plus_2NGHOSTS1 + sj;
  int j = sj;
  __syncthreads();
  
  // fill in SM ghost zones
  if (tid1 < NGHOSTS) {
    uint temp_idx1 = IDX4(UUGF, i, j - NGHOSTS, k);
    s_f[sm_idx - NGHOSTS]  = in_gfs[temp_idx1];
    
    uint temp_idx2 = IDX4(UUGF, i, j + Nxx1, k);
    s_f[sm_idx + Nxx1] = in_gfs[temp_idx2];
  }

  __syncthreads();

  for (int j = tid1 + NGHOSTS; j < Nxx1 + NGHOSTS; j += blockDim.y) {
    int sj = j;
    int sm_idx = si * Nxx_plus_2NGHOSTS1 + sj;

    const REAL uu_j0m2 = s_f[sm_idx - 2];
    const REAL uu_j0m1 = s_f[sm_idx - 1];
    const REAL uu      = s_f[sm_idx    ];
    const REAL uu_j0p1 = s_f[sm_idx + 1];
    const REAL uu_j0p2 = s_f[sm_idx + 2];

    const REAL FDPart1tmp0 = -FDPart1_Rational_5_2 * uu;

    int globalIdx_out = IDX4(UD11, i, j, k);
    aux_gfs[globalIdx_out] = ((invdxx1) * (invdxx1)) * (
        FDPart1_Rational_1_12 * (-uu_j0m2 - uu_j0p2) 
      + FDPart1_Rational_4_3  * ( uu_j0m1 + uu_j0p1) 
      + FDPart1tmp0
    );
    int globalIdx = IDX4(UUGF, i, j, k);
    #ifdef DEBUG_RHS
    if(globalIdx == DEBUG_INDEX) {
      printf("uD11: %1.15f - %1.15f - %1.15f - %1.15f - %1.15f - %1.15f\n", 
        aux_gfs[globalIdx_out], uu_j0m2, uu_j0m1, uu, uu_j0p1, uu_j0p2);
    }
    #endif
  }
}

__global__ void compute_uu_dDDzz_gpu(const params_struct *restrict params, 
                                 const REAL *restrict in_gfs,
                                 REAL *restrict aux_gfs)
{ 

  extern __shared__ REAL s_f[]; // 2-wide halo for 4th order FD

  const REAL & invdxx2 = d_params.invdxx2;

  // const int & Nxx0 = d_params.Nxx0;
  const int & Nxx2 = d_params.Nxx2;

  const int & Nxx_plus_2NGHOSTS0 = d_params.Nxx_plus_2NGHOSTS0;
  const int & Nxx_plus_2NGHOSTS1 = d_params.Nxx_plus_2NGHOSTS1;
  const int & Nxx_plus_2NGHOSTS2 = d_params.Nxx_plus_2NGHOSTS2;

  // Local tile indices - not global thread indicies
  int tid0  = blockIdx.x*blockDim.x + threadIdx.x;
  int tid1  = threadIdx.y;
  int tid2  = blockIdx.y;
  int si = threadIdx.x; // local i for shared memory access

  // Global array indicies
  int i = tid0 + NGHOSTS;
  int j = tid2 + NGHOSTS;

  for (int k = tid1 + NGHOSTS; k < Nxx2 + NGHOSTS; k += blockDim.y) {
    int sk = k;
    
    // s_f stores pencils in linear memory so we need a
    // shared memory index such that the contiguous elements
    // are now the "y" data.
    int sm_idx = si * Nxx_plus_2NGHOSTS2 + sk;

    int globalIdx = IDX4(UUGF, i, j, k);

    s_f[sm_idx] = in_gfs[globalIdx];
  }

  int sk = tid1 + NGHOSTS;
  int sm_idx = si * Nxx_plus_2NGHOSTS2 + sk;
  int k = sk;
  __syncthreads();
  
  // fill in SM ghost zones
  if (tid1 < NGHOSTS) {
    uint temp_idx1 = IDX4(UUGF, i, j, k - NGHOSTS);
    s_f[sm_idx-NGHOSTS]  = in_gfs[temp_idx1];
    uint temp_idx2 = IDX4(UUGF, i, j, k + Nxx2);
    // printf("%d - %d : %d - %d \n", temp_idx1, sm_idx-4, temp_idx2, sm_idx + Nxx1 + NGHOSTS);
    s_f[sm_idx + Nxx2] = in_gfs[temp_idx2];
  }

  __syncthreads();

  for (int k = tid1 + NGHOSTS; k < Nxx2 + NGHOSTS; k += blockDim.y) {
    int sk = k;
    int sm_idx = si * Nxx_plus_2NGHOSTS2 + sk;

    const REAL uu_k0m2 = s_f[sm_idx - 2];
    const REAL uu_k0m1 = s_f[sm_idx - 1];
    const REAL uu      = s_f[sm_idx    ];
    const REAL uu_k0p1 = s_f[sm_idx + 1];
    const REAL uu_k0p2 = s_f[sm_idx + 2];

    const REAL FDPart1tmp0 = -FDPart1_Rational_5_2 * uu;

    int globalIdx_out = IDX4(UD22, i, j, k);
    aux_gfs[globalIdx_out] = ((invdxx2) * (invdxx2)) * (
        FDPart1_Rational_1_12 * (-uu_k0m2 - uu_k0p2) 
      + FDPart1_Rational_4_3  * ( uu_k0m1 + uu_k0p1) 
      + FDPart1tmp0
    );
    int globalIdx = IDX4(UUGF, i, j, k);
    #ifdef DEBUG_RHS
    if(globalIdx == DEBUG_INDEX) {
      printf("uD22: %1.15f - %1.15f - %1.15f - %1.15f - %1.15f - %1.15f\n", 
        aux_gfs[globalIdx_out], uu_k0m2, uu_k0m1, uu, uu_k0p1, uu_k0p2);
    }
    #endif
  }
}

__host__ 
void compute_uu_dDDxx(const params_struct *restrict params, 
                          const REAL *restrict in_gfs,
                          REAL *restrict aux_gfs,
                          const int Nxx0,
                          const int Nxx1,
                          const int Nxx2,
                          const int Nxx_plus_2NGHOSTS0) {
  size_t threads_in_x_dir = Nxx0;
  size_t threads_in_y_dir = 1024 / threads_in_x_dir;
  size_t threads_in_z_dir = 1;

  // Setup our thread layout
  dim3 block_threads(threads_in_x_dir, threads_in_y_dir, threads_in_z_dir);
  dim3 grid_blocks(Nxx1 / threads_in_y_dir, Nxx2, 1);

  // Determine dynamic shared memory size in bytes
  const size_t SM_size = threads_in_y_dir * Nxx_plus_2NGHOSTS0 * sizeof(REAL);
  
  // Fetch maximum shared memory size per block
  const int device = 0; //assumes single GPU
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, device);
  const size_t maxMemPerBlock = deviceProp.sharedMemPerBlock;
  
  if(SM_size > maxMemPerBlock) {
    throw std::runtime_error("Grid is too large for current GPU shared memory restrictions");
  }

  compute_uu_dDDxx_gpu<<<grid_blocks, block_threads, SM_size>>>(params, in_gfs, aux_gfs);
  cudaCheckErrors(compute_uu_dDDxx_gpu, "kernel failed")
}

__host__ 
void compute_uu_dDDyy(const params_struct *restrict params, 
                          const REAL *restrict in_gfs,
                          REAL *restrict aux_gfs,
                          const int Nxx0,
                          const int Nxx1,
                          const int Nxx2,
                          const int Nxx_plus_2NGHOSTS1) {
  const int device = 0; //assumes single GPU
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, device);

  // To ensure coalescence, we want retain reads in the x-direction
  // i.e. the contiguous memory space, based on the standard
  // instruction limits.  Each warp will attempt memory reads up
  // to 128 bytes in a single instruction, in powers of 2,
  // (e.g. 8,16,32,64,128), but this is dependent on the 
  // compute capability of the GPU.  Here we dedicate
  // one thread per data element we read in the x-direction.
  size_t threads_in_x_dir = 128 / sizeof(REAL);

  // Max threads in the y-direction.  Even if we can read
  // the entire tile into shared memory, that doesn't mean
  // we have enough threads per SM to process the entire tile.
  // Therefore we can only have a maximum number of threads in the
  // y direction and each thread will have to compute multiple points.
  size_t threads_in_y_dir = deviceProp.maxThreadsPerBlock / threads_in_x_dir;

  // The tile size should attempt to avoid halo data,
  // i.e. zones of data that are read by two or more blocks
  // into shared memory.
  // For the simple case of cartesian coordinates, the
  // derivatives are 1D, so this shouldn't be a problem
  // so long as the 1D Grid size * threads_in_x_dir
  // will fit into shared memory.
  // Determine dynamic shared memory size in bytes:
  const size_t SM_size = threads_in_x_dir * Nxx_plus_2NGHOSTS1 * sizeof(REAL);
  
  // For now we just throw an exception based on maximum shared memory size per block
  // but it should be possible to decide a better course of action - e.g. using halos
  // instead
  const size_t maxMemPerBlock = deviceProp.sharedMemPerBlock;
  if(SM_size > maxMemPerBlock) {
    throw std::runtime_error("compute_uu_dDDyy: Grid is too large for current GPU shared memory restrictions");
  }

  // Setup our thread layout
  dim3 block_threads(threads_in_x_dir, threads_in_y_dir, 1);
  
  // Setup our grid layout such that our tiles will iterate through the entire
  // numerical space
  dim3 grid_blocks(Nxx0 / threads_in_x_dir, Nxx2, 1);

  // printf("SM: %u - (%u, %u, %u) - (%u, %u, %u)\n", SM_size,
  //   block_threads.x, block_threads.y, block_threads.z, 
  //   grid_blocks.x, grid_blocks.y, grid_blocks.z);
  
  compute_uu_dDDyy_gpu<<<grid_blocks, block_threads, SM_size>>>(params, in_gfs, aux_gfs);
  cudaCheckErrors(compute_uu_dDDyy_gpu, "kernel failed")
  // hipDeviceSynchronize();
  // printf("SM lengh: %u\n",threads_in_x_dir * Nxx_plus_2NGHOSTS1);
}

__host__ 
void compute_uu_dDDzz(const params_struct *restrict params, 
                          const REAL *restrict in_gfs,
                          REAL *restrict aux_gfs,
                          const int Nxx0,
                          const int Nxx1,
                          const int Nxx2,
                          const int Nxx_plus_2NGHOSTS2) {
  const int device = 0; //assumes single GPU
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, device);
  
  // To ensure coalescence, we want retain reads in the x-direction
  // i.e. the contiguous memory space, based on the standard
  // instruction limits.  Each warp will attempt memory reads up
  // to 128 bytes in a single instruction, in powers of 2,
  // (e.g. 8,16,32,64,128), but this is dependent on the 
  // compute capability of the GPU.  Here we dedicate
  // one thread per data element we read in the x-direction.
  size_t threads_in_x_dir = 128 / sizeof(REAL);

  // Max threads in the y-direction.  Even if we can read
  // the entire tile into shared memory, that doesn't mean
  // we have enough threads per SM to process the entire tile.
  // Therefore we can only have a maximum number of threads in the
  // y direction and each thread will have to compute multiple points.
  size_t threads_in_z_dir = deviceProp.maxThreadsPerBlock / threads_in_x_dir;

  // The tile size should attempt to avoid halo data,
  // i.e. zones of data that are read by two or more blocks
  // into shared memory.
  // For the simple case of cartesian coordinates, the
  // derivatives are 1D, so this shouldn't be a problem
  // so long as the 1D Grid size * threads_in_x_dir
  // will fit into shared memory.
  // Determine dynamic shared memory size in bytes:
  const size_t SM_size = threads_in_x_dir * Nxx_plus_2NGHOSTS2 * sizeof(REAL);
  
  // For now we just throw an exception based on maximum shared memory size per block
  // but it should be possible to decide a better course of action - e.g. using halos
  // instead
  const size_t maxMemPerBlock = deviceProp.sharedMemPerBlock;
  if(SM_size > maxMemPerBlock) {
    throw std::runtime_error("compute_uu_dDDzz: Grid is too large for current GPU shared memory restrictions");
  }

  // Setup our thread layout
  dim3 block_threads(threads_in_x_dir, threads_in_z_dir, 1);
  
  // Setup our grid layout such that our tiles will iterate through the entire
  // numerical space
  dim3 grid_blocks(Nxx0 / threads_in_x_dir, Nxx1, 1);

  // printf("SM_size : %lu , max: %lu\n\n\n", SM_size, maxMemPerBlock);
  
  compute_uu_dDDzz_gpu<<<grid_blocks, block_threads, SM_size>>>(params, in_gfs, aux_gfs);
  cudaCheckErrors(compute_uu_dDDzz_gpu, "kernel failed")
}

__global__ void compute_rhs_gpu(const params_struct *restrict params, 
                                 const REAL *restrict in_gfs,
                                 const REAL *restrict in_gfs_derivatives,
                                 REAL *restrict out_gfs)
{ 

  const int & Nxx_plus_2NGHOSTS0 = d_params.Nxx_plus_2NGHOSTS0;
  const int & Nxx_plus_2NGHOSTS1 = d_params.Nxx_plus_2NGHOSTS1;
  const int & Nxx_plus_2NGHOSTS2 = d_params.Nxx_plus_2NGHOSTS2;

  // Local tile indices - not global thread indicies
  int tid0  = threadIdx.x;
  int tid1  = blockIdx.x*blockDim.y + threadIdx.y;
  int tid2  = blockIdx.y;

  // Global memory index - need to shift by ghost zones
  int i = tid0 + NGHOSTS;
  int j = tid1 + NGHOSTS;
  int k = tid2 + NGHOSTS;

  const REAL vv = in_gfs[IDX4(VVGF, i, j, k)];
  const REAL uu_dDD00 = in_gfs_derivatives[IDX4(UD00, i, j, k)];
  const REAL uu_dDD11 = in_gfs_derivatives[IDX4(UD11, i, j, k)];
  const REAL uu_dDD22 = in_gfs_derivatives[IDX4(UD22, i, j, k)];

  const REAL FDPart3tmp0 = ((wavespeed) * (wavespeed));
  out_gfs[IDX4(UUGF, i, j, k)] = vv;
  out_gfs[IDX4(VVGF, i, j, k)] = FDPart3tmp0 * uu_dDD00 + FDPart3tmp0 * uu_dDD11 + FDPart3tmp0 * uu_dDD22;
  #ifdef DEBUG_RHS
  if(IDX4(UUGF, i, j, k) == DEBUG_INDEX) {
    printf("rhs:  %1.15f - %1.15f\n\n",
    out_gfs[IDX4(UUGF, i, j, k)], out_gfs[IDX4(VVGF, i, j, k)]);
  }
  // printf("\ntid1 : %u\n", tid1);
  #endif
}

__host__ 
void compute_rhs(const params_struct *restrict params, 
                          const REAL *restrict in_gfs,
                          const REAL *restrict aux_gfs,
                          REAL *restrict out_gfs,
                          const int Nxx0,
                          const int Nxx1,
                          const int Nxx2) {
  // To ensure coalescence, we want retain reads in the x-direction
  // i.e. the contiguous memory space, based on the standard
  // instruction limits.  Each warp will attempt memory reads up
  // to 128 bytes in a single instruction, in powers of 2,
  // (e.g. 8,16,32,64,128), but this is dependent on the 
  // compute capability of the GPU.  Here we dedicate
  // one thread per data element we read in the x-direction.
  size_t threads_in_x_dir = MIN(1024, Nxx0);

  // Max threads in the y-direction.  Even if we can read
  // the entire tile into shared memory, that doesn't mean
  // we have enough threads per SM to process the entire tile.
  // Therefore we can only have a maximum number of threads in the
  // y direction and each thread will have to compute multiple points.
  size_t threads_in_y_dir = 1024 / threads_in_x_dir;

  size_t threads_in_z_dir = 1;

  // Setup our thread layout
  dim3 block_threads(threads_in_x_dir, threads_in_y_dir, threads_in_z_dir);
  
  // Setup our grid layout such that our tiles will iterate through the entire
  // numerical space
  dim3 grid_blocks(Nxx1 / threads_in_y_dir, Nxx2, 1);

  // printf("SM_size : %lu , max: %lu\n\n\n", SM_size, maxMemPerBlock);
  
  compute_rhs_gpu<<<grid_blocks, block_threads>>>(params, in_gfs, aux_gfs, out_gfs);
  cudaCheckErrors(compute_rhs_gpu, "kernel failed")
}

__host__
void rhs_eval(const commondata_struct *restrict commondata, 
              const params_struct *restrict params, 
              const REAL *restrict in_gfs,
              REAL *restrict rhs_gfs,
              REAL *restrict aux_gfs) {
// #define ORIG_RHS
#ifdef ORIG_RHS
  int Nxx_plus_2NGHOSTS0, Nxx_plus_2NGHOSTS1, Nxx_plus_2NGHOSTS2;
  hipMemcpy(&Nxx_plus_2NGHOSTS0, &params->Nxx_plus_2NGHOSTS0, sizeof(int), hipMemcpyDeviceToHost);
  cudaCheckErrors(hipMemcpy, "memory failed")
  hipMemcpy(&Nxx_plus_2NGHOSTS1, &params->Nxx_plus_2NGHOSTS1, sizeof(int), hipMemcpyDeviceToHost);
  cudaCheckErrors(hipMemcpy, "memory failed")
  hipMemcpy(&Nxx_plus_2NGHOSTS2, &params->Nxx_plus_2NGHOSTS2, sizeof(int), hipMemcpyDeviceToHost);
  cudaCheckErrors(hipMemcpy, "memory failed")
  dim3 block(GPU_NBLOCK0,GPU_NBLOCK1,GPU_NBLOCK2);
  dim3 grid(
    (Nxx_plus_2NGHOSTS0 + GPU_NBLOCK0 - 1) / GPU_NBLOCK0,
    (Nxx_plus_2NGHOSTS1 + GPU_NBLOCK1 - 1) / GPU_NBLOCK1,
    (Nxx_plus_2NGHOSTS2 + GPU_NBLOCK2 - 1) / GPU_NBLOCK2
  );
  rhs_eval_gpu<<<grid,block>>>(commondata, params, in_gfs, rhs_gfs);
  cudaCheckErrors(rhs_eval_gpu, "kernel failed")
  // testcpy(in_gfs);
#else
  // Nxx per coordinate direction
  int Nxx0, Nxx1, Nxx2;
  hipMemcpy(&Nxx0, &params->Nxx0, sizeof(int), hipMemcpyDeviceToHost);
  cudaCheckErrors(hipMemcpy, "memory failed")
  hipMemcpy(&Nxx1, &params->Nxx1, sizeof(int), hipMemcpyDeviceToHost);
  cudaCheckErrors(hipMemcpy, "memory failed")
  hipMemcpy(&Nxx2, &params->Nxx2, sizeof(int), hipMemcpyDeviceToHost);
  cudaCheckErrors(hipMemcpy, "memory failed")
  
  // Nxx + ghost zones per coordinate direction
  int Nxx_plus_2NGHOSTS0, Nxx_plus_2NGHOSTS1, Nxx_plus_2NGHOSTS2;
  hipMemcpy(&Nxx_plus_2NGHOSTS0, &params->Nxx_plus_2NGHOSTS0, sizeof(int), hipMemcpyDeviceToHost);
  cudaCheckErrors(hipMemcpy, "memory failed")
  hipMemcpy(&Nxx_plus_2NGHOSTS1, &params->Nxx_plus_2NGHOSTS1, sizeof(int), hipMemcpyDeviceToHost);
  cudaCheckErrors(hipMemcpy, "memory failed")
  hipMemcpy(&Nxx_plus_2NGHOSTS2, &params->Nxx_plus_2NGHOSTS2, sizeof(int), hipMemcpyDeviceToHost);
  cudaCheckErrors(hipMemcpy, "memory failed")
  
  compute_uu_dDDxx(params, in_gfs, aux_gfs, Nxx0, Nxx1, Nxx2,Nxx_plus_2NGHOSTS0);
  compute_uu_dDDyy(params, in_gfs, aux_gfs, Nxx0, Nxx1, Nxx2,Nxx_plus_2NGHOSTS1);
  compute_uu_dDDzz(params, in_gfs, aux_gfs, Nxx0, Nxx1, Nxx2,Nxx_plus_2NGHOSTS2);

  compute_rhs(params, in_gfs, aux_gfs, rhs_gfs, Nxx0, Nxx1, Nxx2);
#endif
  // compute_rhs(params, in_gfs, aux_gfs, Nxx0, Nxx1, Nxx2,Nxx_plus_2NGHOSTS2);

  // dim3 block(GPU_NBLOCK0,GPU_NBLOCK1,GPU_NBLOCK2);
  // dim3 grid(
  //   (Nxx_plus_2NGHOSTS0 + GPU_NBLOCK0 - 1) / GPU_NBLOCK0,
  //   (Nxx_plus_2NGHOSTS1 + GPU_NBLOCK1 - 1) / GPU_NBLOCK1,
  //   (Nxx_plus_2NGHOSTS2 + GPU_NBLOCK2 - 1) / GPU_NBLOCK2
  // );
  // rhs_eval_gpu<<<grid,block>>>(commondata, params, in_gfs, rhs_gfs);
  // cudaCheckErrors(rhs_eval_gpu, "kernel failed")
}

#include "../BHaH_defines.h"
/*
 * Free Host storage for diagnostics GFs.
 */
__host__ void CUDA__free_host_gfs(MoL_gridfunctions_struct *gridfuncs) {

  hipHostFree(gridfuncs->y_n_gfs);
  cudaCheckErrors(free, "Host-ynFree failed");
  hipHostFree(gridfuncs->diagnostic_output_gfs);
  cudaCheckErrors(free, "Host-non-ynFree failed");
}

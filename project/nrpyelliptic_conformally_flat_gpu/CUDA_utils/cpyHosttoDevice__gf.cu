#include "../BHaH_defines.h"
/*
 * Asynchronously copying a grid function from host to device.
 */
__host__ size_t cpyHosttoDevice__gf(const commondata_struct *restrict commondata, const params_struct *restrict params, const REAL *gf_host,
                                    REAL *gf_gpu, const int host_GF_IDX, const int gpu_GF_IDX) {

  int const Nxx_plus_2NGHOSTS0 = params->Nxx_plus_2NGHOSTS0;
  int const Nxx_plus_2NGHOSTS1 = params->Nxx_plus_2NGHOSTS1;
  int const Nxx_plus_2NGHOSTS2 = params->Nxx_plus_2NGHOSTS2;
  const int Nxx_plus_2NGHOSTS_tot = Nxx_plus_2NGHOSTS0 * Nxx_plus_2NGHOSTS1 * Nxx_plus_2NGHOSTS2;

  size_t streamid = (params->grid_idx + gpu_GF_IDX) % nstreams;
  int offset_gpu = Nxx_plus_2NGHOSTS_tot * gpu_GF_IDX;
  int offset_host = Nxx_plus_2NGHOSTS_tot * host_GF_IDX;
  hipMemcpyAsync(&gf_gpu[offset_host], &gf_host[offset_gpu], sizeof(REAL) * Nxx_plus_2NGHOSTS_tot, hipMemcpyHostToDevice, streams[streamid]);
  cudaCheckErrors(hipMemcpyAsync, "Copy of gf data failed");
  return streamid;
}

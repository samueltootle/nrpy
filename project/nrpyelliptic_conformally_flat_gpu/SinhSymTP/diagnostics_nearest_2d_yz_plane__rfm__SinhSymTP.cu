#include "hip/hip_runtime.h"
#include "../BHaH_defines.h"
#include "../BHaH_function_prototypes.h"
/*
 * Output diagnostic quantities at gridpoints closest to yz plane.
 */
void diagnostics_nearest_2d_yz_plane__rfm__SinhSymTP(commondata_struct *restrict commondata, const params_struct *restrict params,
                                                     REAL *restrict xx[3], MoL_gridfunctions_struct *restrict gridfuncs) {
#include "../set_CodeParameters.h"

  // Unpack gridfuncs struct:
  [[maybe_unused]] const REAL *restrict y_n_gfs = gridfuncs->y_n_gfs;
  [[maybe_unused]] const REAL *restrict auxevol_gfs = gridfuncs->auxevol_gfs;
  [[maybe_unused]] const REAL *restrict diagnostic_output_gfs = gridfuncs->diagnostic_output_gfs;

  // 1D output
  char filename[256];
  sprintf(filename, "out2d-yz-n-%08d.txt", nn);
  FILE *outfile = (nn == 0) ? fopen(filename, "w") : fopen(filename, "a");
  if (!outfile) {
    fprintf(stderr, "Error: Cannot open file %s for writing.\n", filename);
    exit(1);
  }

  // Output data in yz-plane in SinhSymTP coordinates.
  const int numpts_i0 = Nxx0, numpts_i1 = Nxx1, numpts_i2 = 2;
  int i0_pts[numpts_i0], i1_pts[numpts_i1], i2_pts[numpts_i2];
#pragma omp parallel for
  for (int i0 = NGHOSTS; i0 < Nxx0 + NGHOSTS; i0++)
    i0_pts[i0 - NGHOSTS] = i0;
#pragma omp parallel for
  for (int i1 = NGHOSTS; i1 < Nxx1 + NGHOSTS; i1++)
    i1_pts[i1 - NGHOSTS] = i1;
  i2_pts[0] = (int)(NGHOSTS + (1.0 / 4.0) * Nxx2 - 1.0 / 2.0);
  i2_pts[1] = (int)(NGHOSTS + (3.0 / 4.0) * Nxx2 - 1.0 / 2.0);
  // Main loop:
  LOOP_NOOMP(i0_pt, 0, numpts_i0, i1_pt, 0, numpts_i1, i2_pt, 0, numpts_i2) {
    const int i0 = i0_pts[i0_pt], i1 = i1_pts[i1_pt], i2 = i2_pts[i2_pt];
    const int idx3 = IDX3(i0, i1, i2);
    REAL xCart[3];
    xx_to_Cart(commondata, params, xx, i0, i1, i2, xCart);
    {
      const REAL numUU = y_n_gfs[IDX4pt(UUGF, idx3)];
      const REAL log10ResidualH = log10(fabs(diagnostic_output_gfs[IDX4pt(RESIDUAL_HGF, idx3)] + 1e-16));
      fprintf(outfile, "%.15e %.15e %.15e %.15e\n", xCart[1], xCart[2], numUU, log10ResidualH);
    }
  }

  fclose(outfile);
}

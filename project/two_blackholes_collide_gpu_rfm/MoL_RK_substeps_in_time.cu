#include "hip/hip_runtime.h"
#include "BHaH_defines.h"
#include "BHaH_function_prototypes.h"
#include "BHaH_gpu_defines.h"
#include "BHaH_gpu_function_prototypes.h"
#include <unistd.h>

__global__
void rk_substep1_gpu(const REAL *restrict y_n_gfs,
                REAL *restrict y_nplus1_running_total_gfs,
                REAL *restrict k_odd_gfs,
                const REAL *restrict k_even_gfs,
                REAL *restrict auxevol_gfs,
                REAL const dt,
                int const N) {
    int const Nxx_plus_2NGHOSTS0 = d_params.Nxx_plus_2NGHOSTS0;
    int const Nxx_plus_2NGHOSTS1 = d_params.Nxx_plus_2NGHOSTS1;
    int const Nxx_plus_2NGHOSTS2 = d_params.Nxx_plus_2NGHOSTS2;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    constexpr REAL rk_weight = 1./6.;
    constexpr REAL dt_step_factor = 1./2.;

    for(int i=index;i<N;i+=stride) {
        const REAL k_odd_gfsL = k_odd_gfs[i];
        const REAL y_n_gfsL = y_n_gfs[i];
        y_nplus1_running_total_gfs[i] = rk_weight * dt * k_odd_gfsL;
        k_odd_gfs[i] = dt_step_factor *dt * k_odd_gfsL + y_n_gfsL;
    }
}

__host__
void rk_substep1(params_struct *restrict params,
                REAL *restrict y_n_gfs,
                REAL *restrict y_nplus1_running_total_gfs,
                REAL *restrict k_odd_gfs,
                REAL *restrict k_even_gfs,
                REAL *restrict auxevol_gfs, REAL const dt) {
    // Compute optimal grid/block configuration for GPU
    int const Nxx_plus_2NGHOSTS0 = params->Nxx_plus_2NGHOSTS0;
    int const Nxx_plus_2NGHOSTS1 = params->Nxx_plus_2NGHOSTS1;
    int const Nxx_plus_2NGHOSTS2 = params->Nxx_plus_2NGHOSTS2;
    const int N = Nxx_plus_2NGHOSTS0 \
                * Nxx_plus_2NGHOSTS1 \
                * Nxx_plus_2NGHOSTS2 \
                * NUM_EVOL_GFS;
    int block_threads = MIN(GPU_THREADX_MAX, N/32);
    int grid_blocks = (N + block_threads - 1) / block_threads;
    size_t streamid = params->grid_idx % nstreams;

    rk_substep1_gpu<<<grid_blocks, block_threads, 0, streams[streamid]>>>(y_n_gfs,
                                                   y_nplus1_running_total_gfs,
                                                   k_odd_gfs,
                                                   k_even_gfs,
                                                   auxevol_gfs,
                                                   dt, N);
    cudaCheckErrors(rhs_substep1_gpu, "kernel failed")
}

__global__
void rk_substep2_gpu(params_struct *restrict params,
                const REAL *restrict y_n_gfs,
                REAL *restrict y_nplus1_running_total_gfs,
                const REAL *restrict k_odd_gfs,
                REAL *restrict k_even_gfs,
                REAL *restrict auxevol_gfs,
                REAL const dt,
                size_t const N) {
    
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    constexpr REAL rk_weight = 1./3.;
    constexpr REAL dt_step_factor = 1./2.;

    for(int i=index;i<N;i+=stride) {
        const REAL k_even_gfsL = k_even_gfs[i];
        const REAL y_nplus1_running_total_gfsL = y_nplus1_running_total_gfs[i];
        const REAL y_n_gfsL = y_n_gfs[i];
        y_nplus1_running_total_gfs[i] = rk_weight * dt * k_even_gfsL + y_nplus1_running_total_gfsL;
        k_even_gfs[i] = dt_step_factor * dt * k_even_gfsL + y_n_gfsL;
    }
}

__host__
void rk_substep2(params_struct *restrict params,
                REAL *restrict y_n_gfs,
                REAL *restrict y_nplus1_running_total_gfs,
                REAL *restrict k_odd_gfs,
                REAL *restrict k_even_gfs,
                REAL *restrict auxevol_gfs,REAL const dt) {
    // Compute optimal grid/block configuration for GPU
    int const Nxx_plus_2NGHOSTS0 = params->Nxx_plus_2NGHOSTS0;
    int const Nxx_plus_2NGHOSTS1 = params->Nxx_plus_2NGHOSTS1;
    int const Nxx_plus_2NGHOSTS2 = params->Nxx_plus_2NGHOSTS2;

    const int N = Nxx_plus_2NGHOSTS0 \
                * Nxx_plus_2NGHOSTS1 \
                * Nxx_plus_2NGHOSTS2 \
                * NUM_EVOL_GFS;
    int block_threads = MIN(GPU_THREADX_MAX, N/32);
    int grid_blocks = (N + block_threads - 1) / block_threads;
    size_t streamid = params->grid_idx % nstreams;

    rk_substep2_gpu<<<grid_blocks, block_threads, 0, streams[streamid]>>>(params, 
                                                   y_n_gfs,
                                                   y_nplus1_running_total_gfs,
                                                   k_odd_gfs,
                                                   k_even_gfs,
                                                   auxevol_gfs,
                                                   dt, (size_t) N);
    cudaCheckErrors(rhs_substep2_gpu, "kernel failed")
}

__global__
void rk_substep3_gpu(params_struct *restrict params,
                const REAL *restrict y_n_gfs,
                REAL *restrict y_nplus1_running_total_gfs,
                REAL *restrict k_odd_gfs,
                const REAL *restrict k_even_gfs,
                REAL *restrict auxevol_gfs,
                REAL const dt,
                size_t const N) {
    
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    constexpr REAL rk_weight = 1./3.;
    constexpr REAL dt_step_factor = 1.;

    for(int i=index;i<N;i+=stride) {
        const REAL k_odd_gfsL = k_odd_gfs[i];
        const REAL y_nplus1_running_total_gfsL = y_nplus1_running_total_gfs[i];
        const REAL y_n_gfsL = y_n_gfs[i];
        y_nplus1_running_total_gfs[i] =     \
            rk_weight * dt * k_odd_gfsL     \
            + y_nplus1_running_total_gfsL;
        k_odd_gfs[i] = dt_step_factor * dt * k_odd_gfsL + y_n_gfsL;
    }
}

__host__
void rk_substep3(params_struct *restrict params,
                REAL *restrict y_n_gfs,
                REAL *restrict y_nplus1_running_total_gfs,
                REAL *restrict k_odd_gfs,
                REAL *restrict k_even_gfs,
                REAL *restrict auxevol_gfs, REAL const dt) {
    // Compute optimal grid/block configuration for GPU
    int const Nxx_plus_2NGHOSTS0 = params->Nxx_plus_2NGHOSTS0;
    int const Nxx_plus_2NGHOSTS1 = params->Nxx_plus_2NGHOSTS1;
    int const Nxx_plus_2NGHOSTS2 = params->Nxx_plus_2NGHOSTS2;

    const int N = Nxx_plus_2NGHOSTS0 \
                * Nxx_plus_2NGHOSTS1 \
                * Nxx_plus_2NGHOSTS2 \
                * NUM_EVOL_GFS;
    int block_threads = MIN(GPU_THREADX_MAX, N/32);
    int grid_blocks = (N + block_threads - 1) / block_threads;
    size_t streamid = params->grid_idx % nstreams;

    rk_substep3_gpu<<<grid_blocks, block_threads, 0, streams[streamid]>>>(params, 
                                                   y_n_gfs,
                                                   y_nplus1_running_total_gfs,
                                                   k_odd_gfs,
                                                   k_even_gfs,
                                                   auxevol_gfs,
                                                   dt, (size_t) N);
    cudaCheckErrors(rhs_substep3_gpu, "kernel failed")
}

__global__
void rk_substep4_gpu(params_struct *restrict params,
                REAL *restrict y_n_gfs,
                const REAL *restrict y_nplus1_running_total_gfs,
                REAL *restrict k_odd_gfs,
                const REAL *restrict k_even_gfs,
                REAL *restrict auxevol_gfs,
                REAL const dt,
                size_t const N) {
    
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    constexpr REAL dt_step_factor = 1./6.;

    for(int i=index;i<N;i+=stride) {
        const REAL k_even_gfsL = k_even_gfs[i];
        const REAL y_nplus1_running_total_gfsL = y_nplus1_running_total_gfs[i];
        const REAL y_n_gfsL = y_n_gfs[i];
        y_n_gfs[i] = dt_step_factor * dt * k_even_gfsL + y_n_gfsL + y_nplus1_running_total_gfsL;
    }
}

__host__
void rk_substep4(params_struct *restrict params,
                REAL *restrict y_n_gfs,
                REAL *restrict y_nplus1_running_total_gfs,
                REAL *restrict k_odd_gfs,
                REAL *restrict k_even_gfs,
                REAL *restrict auxevol_gfs, REAL const dt) {
    // Compute optimal grid/block configuration for GPU
    int const Nxx_plus_2NGHOSTS0 = params->Nxx_plus_2NGHOSTS0;
    int const Nxx_plus_2NGHOSTS1 = params->Nxx_plus_2NGHOSTS1;
    int const Nxx_plus_2NGHOSTS2 = params->Nxx_plus_2NGHOSTS2;

    const int N = Nxx_plus_2NGHOSTS0 \
                * Nxx_plus_2NGHOSTS1 \
                * Nxx_plus_2NGHOSTS2 \
                * NUM_EVOL_GFS;
    int block_threads = MIN(GPU_THREADX_MAX, N / 32);
    int grid_blocks = (N + block_threads - 1) / block_threads;
    size_t streamid = params->grid_idx % nstreams;

    rk_substep4_gpu<<<grid_blocks, block_threads, 0, streams[streamid]>>>(params, 
                                                   y_n_gfs,
                                                   y_nplus1_running_total_gfs,
                                                   k_odd_gfs,
                                                   k_even_gfs,
                                                   auxevol_gfs,
                                                   dt, (size_t) N);
    cudaCheckErrors(rhs_substep4_gpu, "kernel failed")
}
#include "../BHaH_defines.h"
/*
 * rfm_precompute_free: reference metric precomputed lookup arrays: free
 */
void rfm_precompute_free__rfm__Spherical(const commondata_struct *restrict commondata, const params_struct *restrict params,
                                         rfm_struct *restrict rfmstruct) {
#include "../set_CodeParameters.h"
  hipFree(rfmstruct->f0_of_xx0);
  hipFree(rfmstruct->f1_of_xx1);
  hipFree(rfmstruct->f1_of_xx1__D1);
  hipFree(rfmstruct->f1_of_xx1__DD11);
}

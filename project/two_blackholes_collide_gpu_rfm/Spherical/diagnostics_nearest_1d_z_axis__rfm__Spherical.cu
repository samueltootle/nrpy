#include "hip/hip_runtime.h"
#include "../BHaH_defines.h"
#include "../BHaH_function_prototypes.h"
#include "../BHaH_gpu_defines.h"
#include "../BHaH_gpu_function_prototypes.h"

typedef struct {
  REAL xCart_axis;
  REAL log10HL;
  REAL log10sqrtM2L;
  REAL cfL;
  REAL alphaL;
  REAL trKL;
} data_point_1d_struct;

// qsort() comparison function for 1D output.
static int compare(const void *a, const void *b) {
  REAL l = ((data_point_1d_struct *)a)->xCart_axis;
  REAL r = ((data_point_1d_struct *)b)->xCart_axis;
  return (l > r) - (l < r);
}

/*
 * Output diagnostic quantities at gridpoints closest to z axis.
 */
void diagnostics_nearest_1d_z_axis__rfm__Spherical(commondata_struct *restrict commondata, const params_struct *restrict params, REAL * xx[3],
                                                   MoL_gridfunctions_struct *restrict gridfuncs) {
#include "../set_CodeParameters.h"

  // Unpack gridfuncs struct:
  __attribute_maybe_unused__ const REAL *restrict y_n_gfs = gridfuncs->y_n_gfs;
  __attribute_maybe_unused__ const REAL *restrict auxevol_gfs = gridfuncs->auxevol_gfs;
  __attribute_maybe_unused__ const REAL *restrict diagnostic_output_gfs = gridfuncs->diagnostic_output_gfs;
  __attribute_maybe_unused__ const REAL *restrict k_odd_gfs = gridfuncs->k_odd_gfs;

  // 1D output
  char filename[256];
  sprintf(filename, "out1d-z-conv_factor%.2f-t%08.2f.txt", convergence_factor, time);
  FILE *outfile = (nn == 0) ? fopen(filename, "w") : fopen(filename, "a");
  if (!outfile) {
    fprintf(stderr, "Error: Cannot open file %s for writing.\n", filename);
    exit(1);
  }

  // Output along z-axis in Spherical coordinates.
  const int numpts_i0 = Nxx_plus_2NGHOSTS0, numpts_i1 = 2, numpts_i2 = 1;
  int i0_pts[numpts_i0], i1_pts[numpts_i1], i2_pts[numpts_i2];

  data_point_1d_struct data_points[numpts_i0 * numpts_i1 * numpts_i2];
  int data_index = 0;

  const auto get_diagnostics = [](auto index, const REAL *restrict g_data) {
    REAL h_data;
    hipMemcpy(&h_data, &g_data[index], sizeof(REAL), hipMemcpyDeviceToHost);
    cudaCheckErrors(hipMemcpy, "memory error");
    return h_data;
  };
  const auto xx_to_cart = [&params] (auto const xx0, auto const xx1, auto const xx2, REAL * xCart) {
    const REAL tmp0 = xx0 * sin(xx1);
    xCart[0] = params->Cart_originx + tmp0 * cos(xx2);
    xCart[1] = params->Cart_originy + tmp0 * sin(xx2);
    xCart[2] = params->Cart_originz + xx0 * cos(xx1);
    return xCart;
  };
#pragma omp parallel for
  // for (int i0 = NGHOSTS; i0 < Nxx0 + NGHOSTS; i0++)
  for (int i0 = 0; i0 < Nxx_plus_2NGHOSTS0; i0++)
    i0_pts[i0] = i0;
  i1_pts[0] = (int)(NGHOSTS);
  i1_pts[1] = (int)(-NGHOSTS + Nxx_plus_2NGHOSTS1 - 1);
  i2_pts[0] = (int)(NGHOSTS);
  // Main loop:
  LOOP_NOOMP(i0_pt, 0, numpts_i0, i1_pt, 0, numpts_i1, i2_pt, 0, numpts_i2) {
    const int i0 = i0_pts[i0_pt], i1 = i1_pts[i1_pt], i2 = i2_pts[i2_pt];
    const int idx3 = IDX3(i0, i1, i2);
    REAL xCart[3];
    {
      const REAL xx0 = get_diagnostics(i0, xx[0]);
      const REAL xx1 = get_diagnostics(i1, xx[1]);
      const REAL xx2 = get_diagnostics(i2, xx[2]);
      xx_to_cart(xx0, xx1, xx2, xCart);
    }

    {
      data_point_1d_struct dp1d;
      dp1d.xCart_axis = xCart[2];
      const REAL HL = get_diagnostics(IDX4pt(HGF, idx3), diagnostic_output_gfs);
      dp1d.log10HL = log10(fabs(HL + 1e-16));
      const REAL M2L = get_diagnostics(IDX4pt(MSQUAREDGF, idx3), diagnostic_output_gfs);
      dp1d.log10sqrtM2L = log10(sqrt(M2L) + 1e-16);
      dp1d.cfL = get_diagnostics(IDX4pt(CFGF, idx3), y_n_gfs);
      dp1d.alphaL = get_diagnostics(IDX4pt(ALPHAGF, idx3), y_n_gfs);
      dp1d.trKL = get_diagnostics(IDX4pt(TRKGF, idx3), y_n_gfs);
      data_points[data_index] = dp1d;
      data_index++;
    }
  }

  qsort(data_points, data_index, sizeof(data_point_1d_struct), compare);

  for (int i = 0; i < data_index; i++) {
    fprintf(outfile, "%.15e %.15e %.15e %.15e %.15e %.15e\n", data_points[i].xCart_axis, data_points[i].log10HL, data_points[i].log10sqrtM2L,
            data_points[i].cfL, data_points[i].alphaL, data_points[i].trKL);
  }

  fclose(outfile);
}

#include "../BHaH_defines.h"
#include "../BHaH_gpu_defines.h"
/*
 * rfm_precompute_malloc: reference metric precomputed lookup arrays: malloc
 */
void rfm_precompute_malloc__rfm__Spherical(const commondata_struct *restrict commondata, const params_struct *restrict params,
                                           rfm_struct *restrict rfmstruct) {
  int Nxx_plus_2NGHOSTS0, Nxx_plus_2NGHOSTS1, Nxx_plus_2NGHOSTS2;
  hipMemcpy(&Nxx_plus_2NGHOSTS0, &params->Nxx_plus_2NGHOSTS0, sizeof(int), hipMemcpyDeviceToHost);
  cudaCheckErrors(hipMemcpy, "memory failed")
  hipMemcpy(&Nxx_plus_2NGHOSTS1, &params->Nxx_plus_2NGHOSTS1, sizeof(int), hipMemcpyDeviceToHost);
  cudaCheckErrors(hipMemcpy, "memory failed")
  hipMemcpy(&Nxx_plus_2NGHOSTS2, &params->Nxx_plus_2NGHOSTS2, sizeof(int), hipMemcpyDeviceToHost);
  cudaCheckErrors(hipMemcpy, "memory failed")

  rfmstruct->f0_of_xx0 = (REAL *)malloc(sizeof(REAL) * Nxx_plus_2NGHOSTS0);
  rfmstruct->f1_of_xx1 = (REAL *)malloc(sizeof(REAL) * Nxx_plus_2NGHOSTS1);
  rfmstruct->f1_of_xx1__D1 = (REAL *)malloc(sizeof(REAL) * Nxx_plus_2NGHOSTS1);
  rfmstruct->f1_of_xx1__DD11 = (REAL *)malloc(sizeof(REAL) * Nxx_plus_2NGHOSTS1);
}

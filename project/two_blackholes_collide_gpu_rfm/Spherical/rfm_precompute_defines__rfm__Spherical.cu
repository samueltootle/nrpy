#include "hip/hip_runtime.h"
#include "../BHaH_defines.h"
#include "../BHaH_gpu_defines.h"
/*
 * rfm_precompute_defines: reference metric precomputed lookup arrays: defines
 */

__global__
void rfm_precompute_defines__rfm__Spherical_xx0_gpu(rfm_struct *restrict rfmstruct, const REAL *restrict xx0) {
  for (int i0 = 0; i0 < d_params.Nxx_plus_2NGHOSTS0; i0++) {
    const REAL xx = xx0[i0];
    rfmstruct->f0_of_xx0[i0] = xx;
  }
}

__global__
void rfm_precompute_defines__rfm__Spherical_xx1_gpu(rfm_struct *restrict rfmstruct, const REAL *restrict xx1) {
  for (int i1 = 0; i1 < d_params.Nxx_plus_2NGHOSTS1; i1++) {
    const REAL xx = xx1[i1];
    rfmstruct->f1_of_xx1[i1] = sin(xx);
  }
}

__global__
void rfm_precompute_defines__rfm__Spherical_xx1__D1_gpu(rfm_struct *restrict rfmstruct, const REAL *restrict xx1) {
  for (int i1 = 0; i1 < d_params.Nxx_plus_2NGHOSTS1; i1++) {
    const REAL xx = xx1[i1];
    rfmstruct->f1_of_xx1__D1[i1] = cos(xx);
  }
}

__global__
void rfm_precompute_defines__rfm__Spherical_xx1__DD11_gpu(rfm_struct *restrict rfmstruct) {
  for (int i1 = 0; i1 < d_params.Nxx_plus_2NGHOSTS1; i1++) {
    rfmstruct->f1_of_xx1__DD11[i1] = -rfmstruct->f1_of_xx1[i1];
  }
}

void rfm_precompute_defines__rfm__Spherical(const commondata_struct *restrict commondata, const params_struct *restrict params,
                                            rfm_struct *restrict rfmstruct, REAL * xx[3]) {

  int const& Nxx_plus_2NGHOSTS0 = params->Nxx_plus_2NGHOSTS0;
  int const& Nxx_plus_2NGHOSTS1 = params->Nxx_plus_2NGHOSTS1;
  int const& Nxx_plus_2NGHOSTS2 = params->Nxx_plus_2NGHOSTS2;
  
  dim3 block_threads, grid_blocks;
  auto set_grid_block = [&block_threads, &grid_blocks](auto Nx) {
    size_t tx = MIN(Nx, 1024);
    block_threads = dim3(tx, 1, 1);
    grid_blocks = dim3((Nx + tx - 1)/tx, 1, 1);
  };
  set_grid_block(Nxx_plus_2NGHOSTS0);
  rfm_precompute_defines__rfm__Spherical_xx0_gpu<<<grid_blocks, block_threads, 0, stream1>>>(rfmstruct, xx[0]);
  cudaCheckErrors(rfm_precompute_defines__rfm__Spherical_xx0_gpu, "kernel failed");

  set_grid_block(Nxx_plus_2NGHOSTS1);
  rfm_precompute_defines__rfm__Spherical_xx1_gpu<<<grid_blocks, block_threads, 0, stream2>>>(rfmstruct, xx[1]);
  cudaCheckErrors(rfm_precompute_defines__rfm__Spherical_xx1_gpu, "kernel failed");
  rfm_precompute_defines__rfm__Spherical_xx1__D1_gpu<<<grid_blocks, block_threads, 0, stream3>>>(rfmstruct, xx[1]);
  cudaCheckErrors(rfm_precompute_defines__rfm__Spherical_xx1__D1_gpu, "kernel failed");
  rfm_precompute_defines__rfm__Spherical_xx1__DD11_gpu<<<grid_blocks, block_threads, 0, stream2>>>(rfmstruct);
  cudaCheckErrors(rfm_precompute_defines__rfm__Spherical_xx1__DD11_gpu, "kernel failed");
}

#include "BHaH_defines.h"
#include "BHaH_function_prototypes.h"
#include "BHaH_gpu_defines.h"
#include "BHaH_gpu_function_prototypes.h"

__host__
void mallocHostgrid(const commondata_struct *restrict commondata, 
                    const params_struct *restrict params,
                    griddata_struct *restrict gd_host,
                    const griddata_struct *restrict gd_gpu) {
  int const& Nxx_plus_2NGHOSTS0 = params->Nxx_plus_2NGHOSTS0;
  int const& Nxx_plus_2NGHOSTS1 = params->Nxx_plus_2NGHOSTS1;
  int const& Nxx_plus_2NGHOSTS2 = params->Nxx_plus_2NGHOSTS2;

  // Set up cell-centered Cartesian coordinate grid, centered at the origin.
  gd_host->xx[0] = (REAL*) malloc(sizeof(REAL) * Nxx_plus_2NGHOSTS0);
  gd_host->xx[1] = (REAL*) malloc(sizeof(REAL) * Nxx_plus_2NGHOSTS1);
  gd_host->xx[2] = (REAL*) malloc(sizeof(REAL) * Nxx_plus_2NGHOSTS2);

}

__host__
void cpyDevicetoHost__grid(const commondata_struct *restrict commondata,
                          griddata_struct *restrict gd_host,
                          const griddata_struct *restrict gd_gpu) {
  for (int grid = 0; grid < commondata->NUMGRIDS; grid++) {
    const params_struct *restrict params = &gd_gpu[grid].params;
    int const& Nxx_plus_2NGHOSTS0 = params->Nxx_plus_2NGHOSTS0;
    int const& Nxx_plus_2NGHOSTS1 = params->Nxx_plus_2NGHOSTS1;
    int const& Nxx_plus_2NGHOSTS2 = params->Nxx_plus_2NGHOSTS2;

    mallocHostgrid(commondata, params, gd_host, gd_gpu);

    hipMemcpy(gd_host[grid].xx[0], gd_gpu[grid].xx[0], sizeof(REAL) * Nxx_plus_2NGHOSTS0, hipMemcpyDeviceToHost);
    hipMemcpy(gd_host[grid].xx[1], gd_gpu[grid].xx[1], sizeof(REAL) * Nxx_plus_2NGHOSTS1, hipMemcpyDeviceToHost);
    hipMemcpy(gd_host[grid].xx[2], gd_gpu[grid].xx[2], sizeof(REAL) * Nxx_plus_2NGHOSTS2, hipMemcpyDeviceToHost);
  }
}

__host__
void cpyDevicetoHost__gf(const commondata_struct *restrict commondata,
                        const params_struct *restrict params,
                        REAL *restrict gf_host,
                        const REAL *restrict gf_gpu,
                        const int GF_IDX) {
  int const& Nxx_plus_2NGHOSTS0 = params->Nxx_plus_2NGHOSTS0;
  int const& Nxx_plus_2NGHOSTS1 = params->Nxx_plus_2NGHOSTS1;
  int const& Nxx_plus_2NGHOSTS2 = params->Nxx_plus_2NGHOSTS2;
  const int Nxx_plus_2NGHOSTS_tot = Nxx_plus_2NGHOSTS0 * Nxx_plus_2NGHOSTS1 * Nxx_plus_2NGHOSTS2;
  
  int streamid = (GF_IDX < nstreams) ? GF_IDX : int(GF_IDX / nstreams) - 1;
  int offset = Nxx_plus_2NGHOSTS_tot * GF_IDX;
  hipMemcpyAsync(gf_host, &gf_gpu[offset], sizeof(REAL) * Nxx_plus_2NGHOSTS_tot, hipMemcpyDeviceToHost, streams[GF_IDX]);
}

// __host__
// void MoL_malloc_y_n_gfs__host(const commondata_struct *restrict commondata, const params_struct *restrict params,
//                         MoL_gridfunctions_struct *restrict gf_host) {
// #include "set_CodeParameters.h"
//   const int Nxx_plus_2NGHOSTS_tot = Nxx_plus_2NGHOSTS0 * Nxx_plus_2NGHOSTS1 * Nxx_plus_2NGHOSTS2;
//   const int NUM_DIAG_OUT
//   hipHostMalloc(&gf_host->y_n_gfs, sizeof(REAL) * Nxx_plus_2NGHOSTS_tot * )
//   // hipMalloc(&gridfuncs->y_n_gfs, sizeof(REAL) * NUM_EVOL_GFS * Nxx_plus_2NGHOSTS_tot);

//   // gridfuncs->diagnostic_output_gfs = gridfuncs->y_nplus1_running_total_gfs;
//   // gridfuncs->diagnostic_output_gfs2 = gridfuncs->k_odd_gfs;
// }
#include "hip/hip_runtime.h"
#include "BHaH_defines.h"
#include "BHaH_function_prototypes.h"
#include "BHaH_gpu_defines.h"

__global__
void rk_substep(commondata_struct *restrict commondata, 
                params_struct *restrict params,
                MoL_gridfunctions_struct *restrict gridfuncs,
                REAL rk_weight,
                REAL dt_step_factor) {
    // Set gridfunction aliases from gridfuncs struct
    // y_n gridfunctions
    REAL *restrict y_n_gfs = gridfuncs->y_n_gfs;
    
    // Temporary timelevel & AUXEVOL gridfunctions:
    REAL *restrict y_nplus1_running_total_gfs = gridfuncs->y_nplus1_running_total_gfs;
    REAL *restrict k_odd_gfs = gridfuncs->k_odd_gfs;
    REAL *restrict k_even_gfs = gridfuncs->k_even_gfs;
    REAL *restrict auxevol_gfs = gridfuncs->auxevol_gfs;
    REAL const dt = commondata->dt;

    const int Nxx_plus_2NGHOSTS0 = params->Nxx_plus_2NGHOSTS0;
    const int Nxx_plus_2NGHOSTS1 = params->Nxx_plus_2NGHOSTS1;
    const int Nxx_plus_2NGHOSTS2 = params->Nxx_plus_2NGHOSTS2;
    
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    const int N = Nxx_plus_2NGHOSTS0 \ 
                * Nxx_plus_2NGHOSTS1 \
                * Nxx_plus_2NGHOSTS2 \
                * NUM_EVOL_GFS;
    for(int i=index;i<N;i+=stride) {
        const REAL k_odd_gfsL = k_odd_gfs[i];
        const REAL y_nplus1_running_total_gfsL = y_nplus1_running_total_gfs[i];
        const REAL y_n_gfsL = y_n_gfs[i];
        y_nplus1_running_total_gfs[i] =     \
            rk_weight * dt * k_odd_gfsL     \
            + y_nplus1_running_total_gfsL;
        k_odd_gfs[i] = dt_step_factor * dt * k_odd_gfsL + y_n_gfsL;
    }
}
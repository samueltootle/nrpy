#include "hip/hip_runtime.h"
#include "BHaH_defines.h"
#include "BHaH_gpu_defines.h"
/*
 * Set up cell-centered Cartesian grids.
 */
__global__
void initialize_grid_gpu(params_struct *restrict params, 
                         REAL *restrict xx0,
                         REAL *restrict xx1,
                         REAL *restrict xx2) {

  const int index  = blockIdx.x * blockDim.x + threadIdx.x;
  const int stride = blockDim.x * gridDim.x;
  
    
  // params_struct *restrict params = &griddata[grid].params;
  const REAL xxmin0 = params->xxmin0;
  const REAL xxmin1 = params->xxmin1;
  const REAL xxmin2 = params->xxmin2;

  for (int j = index; j < params->Nxx_plus_2NGHOSTS0; j+=stride)
    xx0[j] = xxmin0 + ((REAL)(j - NGHOSTS) + (1.0 / 2.0)) * params->dxx0;
  for (int j = index; j < params->Nxx_plus_2NGHOSTS1; j+=stride)
    xx1[j] = xxmin1 + ((REAL)(j - NGHOSTS) + (1.0 / 2.0)) * params->dxx1;
  for (int j = index; j < params->Nxx_plus_2NGHOSTS2; j+=stride)
    xx2[j] = xxmin2 + ((REAL)(j - NGHOSTS) + (1.0 / 2.0)) * params->dxx2;
}

void numerical_grids_and_timestep(commondata_struct *restrict commondata, griddata_struct *restrict griddata, bool calling_for_first_time) {
  // Finding dt is trivial for a cartesian grid so we do it in serial
  commondata->dt = 1e30;
  hipDeviceSynchronize();
  for (int grid = 0; grid < commondata->NUMGRIDS; grid++) {
    
    params_struct *restrict params = &griddata[grid].params;
    const REAL convergence_factor = commondata->convergence_factor;
    const REAL xxmin0 = params->xxmin0;
    const REAL xxmin1 = params->xxmin1;
    const REAL xxmin2 = params->xxmin2;
    const REAL xxmax0 = params->xxmax0;
    const REAL xxmax1 = params->xxmax1;
    const REAL xxmax2 = params->xxmax2;

    params->Nxx0 *= convergence_factor;
    params->Nxx1 *= convergence_factor;
    params->Nxx2 *= convergence_factor;

    params->Nxx_plus_2NGHOSTS0 = params->Nxx0 + 2 * NGHOSTS;
    params->Nxx_plus_2NGHOSTS1 = params->Nxx1 + 2 * NGHOSTS;
    params->Nxx_plus_2NGHOSTS2 = params->Nxx2 + 2 * NGHOSTS;

    params->dxx0 = (xxmax0 - xxmin0) / ((REAL)params->Nxx0);
    params->dxx1 = (xxmax1 - xxmin1) / ((REAL)params->Nxx1);
    params->dxx2 = (xxmax2 - xxmin2) / ((REAL)params->Nxx2);

    params->invdxx0 = ((REAL)params->Nxx0) / (xxmax0 - xxmin0);
    params->invdxx1 = ((REAL)params->Nxx1) / (xxmax1 - xxmin1);
    params->invdxx2 = ((REAL)params->Nxx2) / (xxmax2 - xxmin2);

    // Initialize timestepping parameters to zero if this is the first time this function is called.
    if (calling_for_first_time) {
      commondata->nn = 0;
      commondata->nn_0 = 0;
      commondata->t_0 = 0.0;
      commondata->time = 0.0;
      calling_for_first_time=false;
    }
    commondata->dt = MIN(commondata->dt, commondata->CFL_FACTOR * MIN(params->dxx0, MIN(params->dxx1, params->dxx2))); // CFL condition

    // Allocate storage for the discrete grid and ensure it initially resides on the GPU
    hipMallocManaged(&griddata[grid].xx[0], sizeof(REAL) * params->Nxx_plus_2NGHOSTS0);
    cudaCheckErrors(griddata[grid].xx[0], "Malloc failed");
    hipMemPrefetchAsync(griddata[grid].xx[0], sizeof(REAL) * params->Nxx_plus_2NGHOSTS0, 0);
    cudaCheckErrors(griddata[grid].xx[0], "prefetch failed");
    
    hipMallocManaged(&griddata[grid].xx[1], sizeof(REAL) * params->Nxx_plus_2NGHOSTS1);
    cudaCheckErrors(griddata[grid].xx[1], "Malloc failed");
    hipMemPrefetchAsync(griddata[grid].xx[1], sizeof(REAL) * params->Nxx_plus_2NGHOSTS1, 0);
    cudaCheckErrors(griddata[grid].xx[0], "prefetch failed");

    hipMallocManaged(&griddata[grid].xx[2], sizeof(REAL) * params->Nxx_plus_2NGHOSTS2);
    cudaCheckErrors(griddata[grid].xx[2], "Malloc failed");
    hipMemPrefetchAsync(griddata[grid].xx[2], sizeof(REAL) * params->Nxx_plus_2NGHOSTS2, 0);
    cudaCheckErrors(griddata[grid].xx[0], "prefetch failed");
    
    // Initialize grid coordinates
    dim3 block_threads(1024,1,1);
    dim3 grid_blocks(params->Nxx_plus_2NGHOSTS0 + 1024 - 1);
    initialize_grid_gpu<<<grid_blocks, block_threads>>>(params, 
                                                        griddata[grid].xx[0], 
                                                        griddata[grid].xx[1], 
                                                        griddata[grid].xx[2]);
    cudaCheckErrors(initialize_grid_gpu, "initialize failed");
    hipDeviceSynchronize();
  }
}